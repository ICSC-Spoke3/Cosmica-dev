#include "hip/hip_runtime.h"
// ReSharper disable CppUnusedIncludeDirective
#define MAINCU

// .. standard C
#include <cstdio>          // Supplies FILE, stdin, stdout, stderr, and the fprint() family of functions
#include <cstdlib>         // Supplies malloc(), calloc(), and realloc()
#include <unistd.h>         // Supplies EXIT_FAILURE, EXIT_SUCCESS
#include <sys/types.h>      // Typedef shortcuts like uint32_t and uint64_t
#include <sys/time.h>       // supplies time()
#include <span>
#include <numeric>
#include <deque>
#include <ranges>

// .. multi-thread
#include <omp.h>

// math lib
#include <cmath>           // C math library
// .. CUDA specific
#include <hiprand/hiprand_kernel.h>  // CUDA random number device library
#include <hip/hip_runtime.h>   // Device code management by providing implicit initialization, context management, and module management



// .. project specific
#include <spdlog/spdlog.h>
#include <fkYAML/node.hpp>
#include "VariableStructure.cuh"

#ifndef UNIFIED_COMPILE
#include "LoadConfiguration.cuh"
#include "HeliosphericPropagation.cuh"
#include "GenComputation.cuh"
#include "HistoComputation.cuh"
#include "GPUManage.cuh"
#include "Histogram.cuh"

// .. old HelMod code
#include "HelModVariableStructure.cuh"
#include "IOConfiguration.cuh"
#include "DiffusionModel.cuh"
#endif
#include "HelModVariableStructure.cuh"

#include "EventSequence.hpp"


// Track the errors
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define ERR_NoOutputFile "ERROR: output file cannot be open, do you have writing permission?\n"

// Simulation iperparameters definition
#define WARPSIZE 32
#ifndef SetWarpPerBlock
#define SetWarpPerBlock -1                                // number of warp so be submitted -- modify this value to find the best performance



#endif
#define NPARTS 5000
#define NPOS 10
#define RBINS 100

// Debugging variables
#define VERBOSE 1
#define VERBOSE_2 1
#define VERBOSE_LOAD 0
#define INITSAVE 0
#define FINALSAVE 0

// Datas variables
#define MaxCharinFileName   90

// -----------------------------------------------------------------
// ------------  Device Constant Variables declaration -------------
// -----------------------------------------------------------------
__constant__ SimulationConstants_t Constants;


#ifdef UNIFIED_COMPILE
#include "sources/DiffusionModel.cu"
#include "sources/EventSequence.cpp"
#include "sources/GenComputation.cu"
#include "sources/GPUManage.cu"
#include "sources/HeliosphereModel.cu"
#include "sources/HeliosphericPropagation.cu"
#include "sources/IOConfiguration.cu"
#include "sources/HistoComputation.cu"
#include "sources/Histogram.cu"
#include "sources/LoadConfiguration.cu"
#include "sources/MagneticDrift.cu"
#include "sources/SDECoeffs.cu"
#include "sources/SolarWind.cu"
#endif

bool test_and_pop(std::deque<unsigned> &queue, unsigned &val) {
    bool ret;
#pragma omp critical
    {
        if ((ret = !queue.empty())) {
            val = queue.front();
            queue.pop_front();
        }
    }
    return ret;
}

// Main Code
int main(int argc, char *argv[]) {
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    spdlog::set_level(spdlog::level::trace);
    spdlog::info("Simulation started");

    EventSequence BENCHMARK{"Cosmica"};

    const int NGPUs = AvailableGPUs();

    if (NGPUs < 1) {
        spdlog::critical("No CUDA capable devices were detected");
        exit(EXIT_FAILURE);
    }

    hipDeviceProp_t *GPUs_profile = DeviceInfo(NGPUs);
    omp_set_num_threads(NGPUs);

    spdlog::debug("CPU cores: {} (threads used {})", omp_get_num_procs(), omp_get_num_threads());

    SimConfiguration_t SimParameters;

    if (LoadConfigFile(argc, argv, SimParameters, VERBOSE_LOAD) != EXIT_SUCCESS) {
        spdlog::critical("Error while loading simulation parameters");
        exit(EXIT_FAILURE);
    }

    unsigned NParams = SimParameters.simulation_parametrization.Nparams, NPositions = SimParameters.NInitialPositions,
            NIsotopes = SimParameters.simulation_constants.NIsotopes, NRep = SimParameters.Npart;
    unsigned NInstances = NParams * NIsotopes, NPartsPerInstance = NPositions * NRep;
    unsigned NParts = NInstances * NPartsPerInstance;
    spdlog::info("Simulation parameters loaded:");
    spdlog::info("# of instances: {}", NInstances);
    spdlog::info("# particles per instance: {}", NPartsPerInstance);
    spdlog::info("# total particles: {}", NParts);

    auto Results = SimParameters.Results = AllocateResults(SimParameters.NT, NParts);

    std::string init_filename = SimParameters.output_file_name + "_prop_in.txt";
    std::string final_filename = SimParameters.output_file_name + "_prop_out.txt";
    std::string histo_filename = SimParameters.output_file_name + "_R_histo.txt";

    if (std::remove(init_filename.c_str()) != 0 || std::remove(final_filename.c_str()) != 0) {
        spdlog::warn("Error deleting old propagation files or they do not exist");
    } else {
        spdlog::info("Old propagation files deleted successfully");
    }

    if (std::remove(histo_filename.c_str()) != 0) {
        spdlog::warn("Error deleting old histogram files or it does not exist");
    } else {
        spdlog::info("Old histogram files deleted successfully");
    }

#define USE_RIGIDITY_QUEUE
#ifdef USE_RIGIDITY_QUEUE
    auto rig_indexes = std::views::iota(0u, SimParameters.NT);
    std::deque<unsigned> queue{rig_indexes.begin(), rig_indexes.end()};
    // std::deque<unsigned> queue;
    // for (unsigned i = 0; i < SimParameters.NT; ++i) queue.push_back(i);
#endif

#pragma omp parallel
    {
        unsigned cpu_thread_id = omp_get_thread_num();
        unsigned gpu_id = cpu_thread_id % NGPUs;
        HANDLE_ERROR(hipSetDevice(gpu_id));
        unsigned num_cpu_threads = omp_get_num_threads();

        spdlog::debug("CPU Thread {} (of {}) uses CUDA device {}", cpu_thread_id + 1, num_cpu_threads, gpu_id);

        hipDeviceProp_t device_prop = GPUs_profile[gpu_id];
        auto [BLOCKS, THREADS] = GetLaunchConfig(NParts, device_prop);

        EventSequence THREAD_BENCHMARK(fmt::format("Thread {} Benchmarks", cpu_thread_id + 1));

        auto RandStates = AllocateManagedSafe<hiprandStatePhilox4_32_10_t[]>(NParts);
        unsigned long Rnd_seed = SimParameters.RandomSeed == 0
                                     ? getpid() + time(nullptr) + gpu_id
                                     : SimParameters.RandomSeed;
        hipDeviceSynchronize();
        init_rdmgenerator<<<BLOCKS, THREADS>>>(RandStates.get(), Rnd_seed);
        hipDeviceSynchronize();

        THREAD_BENCHMARK.AddEvent("Random State Initialized");

        spdlog::debug("Random Seeds Configured (seed {})", Rnd_seed);

        CopyToConstant(Constants, &SimParameters.simulation_constants);

        ThreadQuasiParticles_t QuasiParts = AllocateQuasiParticles(NParts);

        ThreadIndexes_t indexes = AllocateIndex(NParts);
        for (unsigned p = 0; p < NParams; ++p) {
            for (unsigned i = 0; i < NIsotopes; ++i) {
                for (unsigned o = 0; o < NPositions; ++o) {
                    for (unsigned x = 0; x < NRep; ++x) {
                        unsigned idx = x + NRep * (o + NPositions * (i + NIsotopes * p));
                        indexes.param[idx] = p;
                        indexes.isotope[idx] = i;
                        indexes.period[idx] = o;
                    }
                }
            }
        }

        THREAD_BENCHMARK.AddEvent("Shared Data Allocated");

#ifdef USE_RIGIDITY_QUEUE
        unsigned iR;
        while (test_and_pop(queue, iR)) {
#else
        for (unsigned int iR = gpu_id; iR < SimParameters.NT; iR += NGPUs) {
#endif
            spdlog::info("Simulation for rigidity {} [{}] started", SimParameters.Tcentr[iR], iR);

            THREAD_BENCHMARK.StartSubsequence(fmt::format("Rigidity {:.3e} [{:02}]", SimParameters.Tcentr[iR], iR));

            for (unsigned iPart = 0; iPart < NParts; ++iPart) {
                QuasiParts.r[iPart] = SimParameters.InitialPositions.r[indexes.period[iPart]];
                QuasiParts.th[iPart] = SimParameters.InitialPositions.th[indexes.period[iPart]];
                QuasiParts.phi[iPart] = SimParameters.InitialPositions.phi[indexes.period[iPart]];
                QuasiParts.R[iPart] = SimParameters.Tcentr[iR]; // TODO: dynamic rigidity based on isotope
                QuasiParts.t_fly[iPart] = 0;
            }


            auto Maxs = AllocateManagedSafe<float[]>(NInstances);
            auto Nfailed = AllocateManagedSafe<unsigned[]>(NInstances, 0);

            THREAD_BENCHMARK.AddEvent("Particles Data Allocated");

            if constexpr (INITSAVE) {
                SaveTxt_part(init_filename.c_str(), NParts, QuasiParts, Maxs[0]);
                THREAD_BENCHMARK.AddEvent("Initial State Stored");
            }


            hipDeviceSynchronize();
            HeliosphericProp<<<BLOCKS, THREADS>>>(QuasiParts, indexes, SimParameters.simulation_parametrization,
                                                  RandStates.get(), Maxs.get());
            hipDeviceSynchronize();

            THREAD_BENCHMARK.AddEvent("Propagation Completed");

            if constexpr (FINALSAVE) {
                SaveTxt_part(final_filename.c_str(), NParts, QuasiParts, Maxs[0]);
                THREAD_BENCHMARK.AddEvent("Final State Stored");
            }


            THREAD_BENCHMARK.StartSubsequence("Histograms Allocation");
            for (unsigned inst = 0; inst < NInstances; ++inst) {
                spdlog::debug("Results for Instance {} (Rigidity {}):", inst, iR);
                spdlog::debug("* R_min: {}, R_max: {}", SimParameters.Tcentr[iR], Maxs[0]);

                if (Maxs[inst] < SimParameters.Tcentr[iR]) {
                    spdlog::error("The max exiting rigidity is smaller than initial one (Instance {})", inst);
                    continue; //TODO: check if needed
                }

                float DeltaLogR = log10f(1.f + SimParameters.RelativeBinAmplitude);
                float LogBin0_lowEdge = log10f(SimParameters.Tcentr[iR]) - DeltaLogR / 2.f;
                float Bin0_lowEdge = powf(10, LogBin0_lowEdge);

                Results[iR][inst].Nbins = ceil(log10(Maxs[inst] / Bin0_lowEdge) / DeltaLogR);
                Results[iR][inst].LogBin0_lowEdge = LogBin0_lowEdge;
                Results[iR][inst].DeltaLogR = DeltaLogR;

                Results[iR][inst].BoundaryDistribution = AllocateManaged<float[]>(Results[iR][inst].Nbins, 0);
                THREAD_BENCHMARK.AddEvent(fmt::format("Histogram {} Allocated", inst));
            }
            THREAD_BENCHMARK.StopSubsequence();

            hipDeviceSynchronize();
            SimpleHistogram<<<BLOCKS, THREADS>>>(indexes, QuasiParts.R, Results[iR], Nfailed.get());
            hipDeviceSynchronize();

            for (unsigned inst = 0; inst < NInstances; ++inst) {
                Results[iR][inst].Nregistered = NPartsPerInstance - Nfailed[inst];
                spdlog::debug("* Total Events.   : {}", NPartsPerInstance);
                spdlog::debug("* Recorded Events : {}", Nfailed[inst]);
                spdlog::debug("* Failed Events.  : {}", Results[iR][inst].Nregistered);
            }

            THREAD_BENCHMARK.AddEvent("Histograms Generated");

            THREAD_BENCHMARK.StopSubsequence();

            spdlog::info("Simulation for rigidity {} [{}] ended", SimParameters.Tcentr[iR], iR);
        }
        // end of the cycle on the rigidities

        if (spdlog::get_level() <= spdlog::level::debug) THREAD_BENCHMARK.Log(spdlog::level::debug, 2);
        else THREAD_BENCHMARK.Log(spdlog::level::info, 1);
    }
    // end of the multiple CPU thread pragma


    ////////////////////////////////////////////////////////////////
    //..... Exit results saving   ..................................
    ////////////////////////////////////////////////////////////////

    // Generate the YAML file name, following the old naming convention:
    std::string yamlFilename = fmt::format("{}_matrix_{}.yaml", SimParameters.output_file_name, getpid());

    try {
        write_results_yaml(yamlFilename, SimParameters);
        spdlog::info("Results saved to file: {}", yamlFilename);
    } catch (const std::exception &e) {
        std::cerr << "Error writing results to YAML file: " << e.what() << std::endl;
        return 1;
    }


    //  Save the summary histogram
    //  Free the dynamic memory

    // Save the rigidity histograms to txt file
    for (unsigned iR = 0; iR < SimParameters.NT; ++iR) {
        SaveTxt_histo(histo_filename.c_str(), Results[iR][0].Nbins, Results[iR][0]);
    }

    /* save results to file .dat */
    FILE *pFile_Matrix = nullptr;
    std::string datFilename = fmt::format("{}_matrix_{}.dat", SimParameters.output_file_name, getpid());

    spdlog::debug("Writing Output File: {}", datFilename);
    pFile_Matrix = fopen(datFilename.c_str(), "w");

    if (pFile_Matrix == nullptr) {
        spdlog::critical("Error, no output file");
        exit(EXIT_FAILURE);
    }

    fprintf(pFile_Matrix, "# COSMICA \n");
    if constexpr (VERBOSE) fprintf(pFile_Matrix, "# Number of Input energies;\n");
    fprintf(pFile_Matrix, "%d \n", SimParameters.NT);

    for (unsigned itemp = 0; itemp < SimParameters.NT; ++itemp) {
        if constexpr (VERBOSE) {
            fprintf(pFile_Matrix, "######  Bin %d \n", itemp);
            fprintf(pFile_Matrix,
                    "# Egen, Npart Gen., Npart Registered, Nbin output, log10(lower edge bin 0), Bin amplitude (in log scale)\n");
        }

        fprintf(pFile_Matrix, "%f %u %u %d %f %f \n", SimParameters.Tcentr[itemp],
                SimParameters.Npart,
                Results[itemp][0].Nregistered,
                Results[itemp][0].Nbins,
                Results[itemp][0].LogBin0_lowEdge,
                Results[itemp][0].DeltaLogR);
        if constexpr (VERBOSE) fprintf(pFile_Matrix, "# output distribution \n");

        for (int itNB = 0; itNB < Results[itemp][0].Nbins; itNB++) {
            fprintf(pFile_Matrix, "%e ", Results[itemp][0].BoundaryDistribution[itNB]);
        }


        fprintf(pFile_Matrix, "\n");
        fprintf(pFile_Matrix, "#\n"); // <--- dummy line to separate results
    }

    fflush(pFile_Matrix);
    fclose(pFile_Matrix);

    delete[] SimParameters.InitialPositions.r;
    delete[] SimParameters.InitialPositions.th;
    delete[] SimParameters.InitialPositions.phi;
    delete[] SimParameters.Tcentr;

    delete[] GPUs_profile;

    BENCHMARK.AddEvent("End");
    BENCHMARK.Log(spdlog::level::info, 0);
    spdlog::info("Simulation ended");

    return hipDeviceReset();
}
