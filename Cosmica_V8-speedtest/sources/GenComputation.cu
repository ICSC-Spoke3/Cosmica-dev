#include "hip/hip_runtime.h"
#include "GenComputation.cuh"
#include "HelModVariableStructure.cuh"

#include <cmath>           // c math library

////////////////////////////////////////////////////////////////
//..... GEneric useful and safe function .......................
////////////////////////////////////////////////////////////////

int ceil_int(const int a, const int b) {
    // https://www.reddit.com/r/C_Programming/comments/gqpuef/comment/fru7tmu/?utm_source=share&utm_medium=web2x&context=3
    return (a + (b - 1)) / b;
}

int floor_int(const int a, const int b) {
    return static_cast<int>(floor(a / b));
}

__device__ float safeSign(const float num) {
    if (num >= 0) return 1;
    return -1;
}

__device__ float SmoothTransition(const float InitialVal, const float FinalVal, const float CenterOfTransition,
                                  const float smoothness,
                                  const float x) {
    if (smoothness == 0) {
        if (x >= CenterOfTransition) return FinalVal;
        return InitialVal;
    }
    return (InitialVal + FinalVal) / 2.f - (InitialVal - FinalVal) / 2.f * tanhf(
               (x - CenterOfTransition) / smoothness);
}

__device__ float beta_(const float T, const float T0) {
    return sqrtf(T * (T + T0 + T0)) / (T + T0);
}

__device__ float beta_R(const float R, const PartDescription_t part) {
    // float T = Energy(R, part);
    // return beta_(T, part.T0);
    return R / sqrtf(sq(R) + part.A * part.A / (part.Z * part.Z) * (part.T0 * part.T0));
}

__device__ __host__ float Rigidity(const float T, const PartDescription_t part) {
    return part.A / fabsf(part.Z) * sqrtf(T * (T + 2.f * part.T0));
}

__device__ __host__ float Energy(const float R, const PartDescription_t part) {
    return sqrtf(part.Z * part.Z / (part.A * part.A) * sq(R) + part.T0 * part.T0) - part.T0;
}
