#include "hip/hip_runtime.h"
#include "GenComputation.cuh"
#include "HelModVariableStructure.cuh"

#include "HeliosphereModel.cuh"
#include "SolarWind.cuh"

/**
 * @brief Calculate the solar wind speed at a given position in the heliosphere.
 *
 * @param index
 * @param r Radial distance
 * @param th th
 * @param phi phi
 * @param LIM
 * @return Solar wind speed
 */
__device__ float SolarWindSpeed(const Index_t &index, const float r,
                                const float th, const float phi, const HeliosphereZoneProperties_t *LIM) {
    const float V0 = index.radial < Heliosphere.Nregions ? LIM[index.combined()].V0 : HS[index.period].V0;


    // heliosheat (or near to)...............................
    if (const float RtsDirection = Boundary(th, phi, Heliosphere.RadBoundary_effe[index.period].Rts_nose,
                                            Heliosphere.RadBoundary_effe[index.period].Rts_tail);
        index.radial >= Heliosphere.Nregions - 1 && r > RtsDirection - L_tl) {
        const float RtsRWDirection = Boundary(th, phi, Heliosphere.RadBoundary_real[index.period].Rts_nose,
                                              Heliosphere.RadBoundary_real[index.period].Rts_tail);
        float DecreasFactor = SmoothTransition(1.f, 1.f / s_tl, RtsDirection, L_tl, r);
        if (r > RtsDirection) {
            DecreasFactor *= sq(RtsRWDirection / (RtsRWDirection - RtsDirection + r));
        }
        return V0 * DecreasFactor;
    }

    // inner Heliosphere .........................
    if (Heliosphere.IsHighActivityPeriod[index.period]) {
        // high solar activity
        return V0;
    }

    return min(Vhigh, V0 * (1 + fabsf(cosf(th))));
}

/**
 * @brief Derivative of solar wind speed in d theta
 *
 * @param index
 * @param r Radial distance
 * @param th th
 * @param phi phi
 * @param LIM
 * @return Derivative of solar wind speed in d theta
 */
__device__ float DerivativeOfSolarWindSpeed_dtheta(const Index_t &index,
                                                   const float r, const float th, const float phi, const HeliosphereZoneProperties_t *LIM) {
    const float V0 = index.radial < Heliosphere.Nregions ? LIM[index.combined()].V0 : HS[index.period].V0;

    // heliosheat ...............................
    // inner Heliosphere .........................
    if (const float RtsDirection = Boundary(th, phi, Heliosphere.RadBoundary_effe[index.period].Rts_nose,
                                            Heliosphere.RadBoundary_effe[index.period].Rts_tail);
        (index.radial >= Heliosphere.Nregions - 1 && r > RtsDirection - L_tl) ||
        Heliosphere.IsHighActivityPeriod[index.period] ||
        V0 * (1 + fabsf(cosf(th))) > Vhigh
    ) {
        return 0;
    }

    return -sign(cosf(th)) * V0 * sinf(th);
}
