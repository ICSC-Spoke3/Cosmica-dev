#include "hip/hip_runtime.h"
#include "GenComputation.cuh"
#include "HelModVariableStructure.cuh"

#include "HeliosphereModel.cuh"
#include "SolarWind.cuh"

/**
 * @brief Calculate the solar wind speed at a given position in the heliosphere.
 *
 * @param InitZone Index of the initial zone
 * @param HZone Index of the heliosphere region
 * @param r Radial distance
 * @param th th
 * @param phi phi
 * @param LIM
 * @return Solar wind speed
 */
__device__ float SolarWindSpeed(const unsigned int InitZone, const signed int HZone, const float r, const float th,
                                const float phi, const HeliosphereZoneProperties_t *LIM) {
    const float V0 = HZone < Heliosphere.Nregions ? LIM[HZone + InitZone].V0 : HS[InitZone].V0;


    // heliosheat (or near to)...............................
    if (const float RtsDirection = Boundary(th, phi, Heliosphere.RadBoundary_effe[InitZone].Rts_nose,
                                            Heliosphere.RadBoundary_effe[InitZone].Rts_tail);
        HZone >= Heliosphere.Nregions - 1 && r > RtsDirection - L_tl) {
        const float RtsRWDirection = Boundary(th, phi, Heliosphere.RadBoundary_real[InitZone].Rts_nose,
                                              Heliosphere.RadBoundary_real[InitZone].Rts_tail);
        float DecreasFactor = SmoothTransition(1.f, 1.f / s_tl, RtsDirection, L_tl, r);
        if (r > RtsDirection) {
            DecreasFactor *= sq(RtsRWDirection / (RtsRWDirection - RtsDirection + r));
        }
        return V0 * DecreasFactor;
    }

    // inner Heliosphere .........................
    if (Heliosphere.IsHighActivityPeriod[InitZone]) {
        // high solar activity
        return V0;
    }

    return min(Vhigh, V0 * (1 + fabsf(cosf(th))));
}

/**
 * @brief Derivative of solar wind speed in d theta
 *
 * @param InitZone Index of the initial zone
 * @param HZone Index of the heliosphere region
 * @param r Radial distance
 * @param th th
 * @param phi phi
 * @param LIM
 * @return Derivative of solar wind speed in d theta
 */
__device__ float DerivativeOfSolarWindSpeed_dtheta(const unsigned int InitZone, const signed int HZone, const float r,
                                                   const float th, const float phi, const HeliosphereZoneProperties_t *LIM) {
    const float V0 = HZone < Heliosphere.Nregions ? LIM[HZone + InitZone].V0 : HS[InitZone].V0;

    // heliosheat ...............................
    // inner Heliosphere .........................
    if (const float RtsDirection = Boundary(th, phi, Heliosphere.RadBoundary_effe[InitZone].Rts_nose,
                                            Heliosphere.RadBoundary_effe[InitZone].Rts_tail);
        (HZone >= Heliosphere.Nregions - 1 && r > RtsDirection - L_tl) ||
        Heliosphere.IsHighActivityPeriod[InitZone] ||
        V0 * (1 + fabsf(cosf(th))) > Vhigh
    ) {
        return 0;
    }

    return -sign(cosf(th)) * V0 * sinf(th);
}
