#include "hip/hip_runtime.h"
#include "GenComputation.cuh"
#include "HeliosphereModel.cuh"
#include "VariableStructure.cuh"
#include "HelModVariableStructure.cuh"
#include <cstdio>          // Supplies FILE, stdin, stdout, stderr, and the fprint() family of functions


/////////////////////////////////////////////////////////////////////////////
////////////////// Diffusion Parameters Description /////////////////////////
/////////////////////////////////////////////////////////////////////////////
void RescaleToEffectiveHeliosphere(HeliosphereBoundRadius_t &Rbound, vect3D_t &part) {
    /* * description: create an effective heliosphere of 100 AU. thisis due to the fact that K0 parameters are tuned on such dimension.
         \param  Rbound heliospher boundaries to be rescaled
         \param  part   initial position to be rescaled
     */
    const float Rts_nose_realworld = Rbound.Rts_nose;
    const float Rhp_nose_realworld = Rbound.Rhp_nose;
    const float Rts_tail_realworld = Rbound.Rts_tail;
    const float Rhp_tail_realworld = Rbound.Rhp_tail;

    Rbound.Rts_nose = 100.;
    Rbound.Rts_tail = Rts_tail_realworld * Rbound.Rts_nose / Rts_nose_realworld;

    Rbound.Rhp_nose = Rbound.Rts_nose + (Rhp_nose_realworld - Rts_nose_realworld); //122.;
    Rbound.Rhp_tail = Rbound.Rts_tail + (Rhp_tail_realworld - Rts_tail_realworld); //Rhp_tail*Rhp/Rhp_realworld;

    const float HM_Rts_d = Boundary(part.th, part.phi, Rbound.Rts_nose, Rbound.Rts_tail);
    const float RW_Rts_d = Boundary(part.th, part.phi, Rts_nose_realworld, Rts_tail_realworld);
    if (const float Rdi_real = part.r; Rdi_real <= RW_Rts_d) part.r = Rdi_real / RW_Rts_d * HM_Rts_d;
    else part.r = HM_Rts_d + (Rdi_real - RW_Rts_d);
}


float K0Fit_ssn(const int p, const int SolarPhase, const float ssn, float *GaussVar) {
    /*Authors: 2011 Stefano - update 2012 Stefano - update 2015 GLV*/
    /* * description: K0 evaluated using ssn as a proxy
         \param p            solar polarity of HMF
         \param SolarPhase   0=rising / 1=Declining phase of solar activity cycle
         \param ssn          smoothed sunspot number
         \param *GaussVar    gaussian variation (output)
     */
    float k0;
    if (p > 0.) {
        if (SolarPhase == 0)/*Rising*/ {
            k0 = 0.0002743f - 2.11e-6f * ssn + 1.486e-8f * sq(ssn) - 3.863e-11f * sq(ssn) * ssn;
            *GaussVar = 0.1122;
        } else /*Declining*/ {
            k0 = 0.0002787f - 1.66e-6f * ssn + 4.658e-9f * sq(ssn) - 6.673e-12f * sq(ssn) * ssn;
            *GaussVar = 0.1324f;
        }
    } else {
        if (SolarPhase == 0)/*Rising*/ {
            k0 = 0.0003059f - 2.51e-6f * ssn + 1.284e-8f * sq(ssn) - 2.838e-11f * sq(ssn) * ssn;
            *GaussVar = 0.1097;
        } else /*Declining*/ {
            k0 = 0.0002876f - 3.715e-6f * ssn + 2.534e-8f * sq(ssn) - 5.689e-11f * sq(ssn) * ssn;
            *GaussVar = 0.14;
        }
    }
    return k0;
}

float K0Fit_NMC(const float NMC, float *GaussVar) {
    /*Authors: 2015 GLV*/
    /* * description: K0 evaluated using Mc Murdo NM counts as a proxy
                      only for High Activity, defined as Tilt L >48deg
       \param NMC          Neutron monitor counting rate from Mc Murdo
       \param *GaussVar    gaussian variation (output)
    */
    *GaussVar = 0.1045;
    return expf(-10.83f - 0.0041f * NMC + 4.52e-5f * sq(NMC));
}

float K0CorrFactor(const int p, const int q, const int SolarPhase, const float tilt) {
    /*Authors: 2017 Stefano */
    /* * description: Correction factor to K0 for the Kparallel. This correction is introduced
                      to account for the fact that K0 is evaluated with a model not including particle drift.
                      Thus, the value need a correction once to be used in present model
        \param p            solar polarity of HMF
        \param q            signum of particle charge
        \param SolarPhase   0=rising / 1=Declining phase of solar activity cycle
        \param tilt         Tilt angle of neutral sheet (in degree)
    */
#ifndef K0Corr_maxv
#define K0Corr_maxv 1.5f
#endif
#ifndef K0Corr_minv
#define K0Corr_minv 1.f
#endif
#ifndef K0Corr_p0_asc
#define K0Corr_p0_asc 18.f
#endif
#ifndef K0Corr_p1_asc
#define K0Corr_p1_asc 40.f
#endif
#ifndef K0Corr_p0_des
#define K0Corr_p0_des 5.f
#endif
#ifndef K0Corr_p1_des
#define K0Corr_p1_des 53.f
#endif
#ifndef K0Corr_maxv_neg
#define K0Corr_maxv_neg 0.7f
#endif
#ifndef K0Corr_p0_asc_neg
#define K0Corr_p0_asc_neg 5.8f
#endif
#ifndef K0Corr_p1_asc_neg
#define K0Corr_p1_asc_neg 47.f
#endif
#ifndef K0Corr_p0_des_neg
#define K0Corr_p0_des_neg 5.8f
#endif
#ifndef K0Corr_p1_des_neg
#define K0Corr_p1_des_neg 58.f
#endif

    if (q > 0) {
        if (q * p > 0) {
            if (SolarPhase == 0) {
                //ascending
                return SmoothTransition(K0Corr_maxv, K0Corr_minv, K0Corr_p1_asc, K0Corr_p0_asc, tilt);
            }
            //descending
            return SmoothTransition(K0Corr_maxv, K0Corr_minv, K0Corr_p1_des, K0Corr_p0_des, tilt);
        }
        return 1;
    }
    if (q < 0) {
        if (q * p > 0) {
            if (SolarPhase == 0) {
                //ascending
                return SmoothTransition(K0Corr_maxv, K0Corr_minv, K0Corr_p1_asc, K0Corr_p0_asc, tilt);
            }
            //descending
            return SmoothTransition(K0Corr_maxv, K0Corr_minv, K0Corr_p1_des, K0Corr_p0_des, tilt);
        }
        if (SolarPhase == 0) {
            //ascending
            return SmoothTransition(K0Corr_maxv_neg, K0Corr_minv, K0Corr_p1_asc_neg, K0Corr_p0_asc_neg, tilt);
        }
        //descending
        return SmoothTransition(K0Corr_maxv_neg, K0Corr_minv, K0Corr_p1_des_neg, K0Corr_p0_des_neg, tilt);
    }
    return 1;
}


float3 EvalK0(const bool IsHighActivityPeriod, const int p, const int q, const int SolarPhase, const float tilt,
              const float NMC, const float ssn,
              const unsigned char verbose = 0) {
    /*Authors: 2022 Stefano */
    /* * description: Evaluate diffusion parameter from fitting procedures.
        \param p            solar polarity of HMF
        \param q            signum of particle charge
        \param SolarPhase   0=rising / 1=Declining phase of solar activity cycle
        \param tilt         Tilt angle of neutral sheet (in degree)
        \return x = k0_paral
                y = k0_perp
                z = GaussVar
    */
    float3 output;
    output.x = K0CorrFactor(p, q, SolarPhase, tilt); // k0_paral is corrected by a correction factor
    ////////////////////////////////////////////////
    // printf("-- p: %d q: %d phase: %d tilt: %e ssn: %e NMC: %e \n",p,q,SolarPhase,tilt,ssn,NMC);
    // printf("-- K0CorrF: %e \n",output.x);
    // printf("-- IsHighActivityPeriod %d \n",IsHighActivityPeriod);
    ////////////////////////////////////////////////
    if (IsHighActivityPeriod && NMC > 0) {
        output.y = K0Fit_NMC(NMC, &output.z);
        output.x *= output.y;
    } else {
        if (verbose >= VERBOSE_med && IsHighActivityPeriod && NMC == 0) {
            fprintf(
                stderr,
                "WARNING:: High Activity period require NMC variable setted with value >0, used ssn instead.\n");
        }
        output.y = K0Fit_ssn(p, SolarPhase, ssn, &output.z);
        output.x *= output.y;
    }
    ////////////////////////////////////////////////
    // printf("-- K0 paral: %e \n",output.x);
    // printf("-- K0 perp : %e \n",output.y);
    ////////////////////////////////////////////////
    return output;
}

float g_low(const int SolarPhase, const int Polarity, const float tilt) {
    /*Authors: 2022 Stefano - update 2023 GLV*/
    /* * description: evaluate g_low parameter (for Kparallel).
        \param SolarPhase   0=rising / 1=Declining phase of solar activity cycle
        \param tilt         Tilt angle of neutral sheet (in degree)
        \return g_low
    */
#ifndef MaxValueOf_g_low_pos
#define MaxValueOf_g_low_pos 0.6f
#endif
#ifndef CAB_TransPoint_des_pos
#define CAB_TransPoint_des_pos 45
#endif
#ifndef CAB_smoothness_des_pos
#define CAB_smoothness_des_pos 5.f
#endif
#ifndef CAB_TransPoint_asc_pos
#define CAB_TransPoint_asc_pos 60
#endif
#ifndef CAB_smoothness_asc_pos
#define CAB_smoothness_asc_pos 9.f
#endif
#ifndef MaxValueOf_g_low_neg
#define MaxValueOf_g_low_neg 0.5f
#endif
#ifndef CAB_TransPoint_des_neg
#define CAB_TransPoint_des_neg 45
#endif
#ifndef CAB_smoothness_des_neg
#define CAB_smoothness_des_neg 10.f
#endif
#ifndef CAB_TransPoint_asc_neg
#define CAB_TransPoint_asc_neg 60.f
#endif
#ifndef CAB_smoothness_asc_neg
#define CAB_smoothness_asc_neg 9.f
#endif
    float g_low = 0;
    float MaxValueOf_g_low, CAB_TransPoint_des, CAB_smoothness_des, CAB_TransPoint_asc, CAB_smoothness_asc;
    if (Polarity > 0) {
        MaxValueOf_g_low = MaxValueOf_g_low_pos;
        CAB_TransPoint_des = CAB_TransPoint_des_pos;
        CAB_smoothness_des = CAB_smoothness_des_pos;
        CAB_TransPoint_asc = CAB_TransPoint_asc_pos;
        CAB_smoothness_asc = CAB_smoothness_asc_pos;
    } else {
        MaxValueOf_g_low = MaxValueOf_g_low_neg;
        CAB_TransPoint_des = CAB_TransPoint_des_neg;
        CAB_smoothness_des = CAB_smoothness_des_neg;
        CAB_TransPoint_asc = CAB_TransPoint_asc_neg;
        CAB_smoothness_asc = CAB_smoothness_asc_neg;
    }

    if (SolarPhase == 1) {
        g_low = MaxValueOf_g_low * SmoothTransition(1, 0, CAB_TransPoint_des, CAB_smoothness_des, tilt);
    } else {
        g_low = MaxValueOf_g_low * SmoothTransition(1, 0, CAB_TransPoint_asc, CAB_smoothness_asc, tilt);
    }
    return g_low;
}

float rconst(const int SolarPhase, const int Polarity, const float tilt) {
    /*Authors: 2022 Stefano - update 2023 GLV*/
    /* * description: evaluate rconst parameter (for Kparallel).
        \param SolarPhase   0=rising / 1=Declining phase of solar activity cycle
        \param tilt         Tilt angle of neutral sheet (in degree)
        \return rconst
    */
#ifndef MaxValueOf_rconst
#define MaxValueOf_rconst 4
#endif
#ifndef rconst_TransPoint_des_pos
#define rconst_TransPoint_des_pos 45
#endif
#ifndef rconst_smoothness_des_pos
#define rconst_smoothness_des_pos 5.f
#endif
#ifndef rconst_TransPoint_asc_pos
#define rconst_TransPoint_asc_pos 60
#endif
#ifndef rconst_smoothnesst_asc_pos
#define rconst_smoothness_asc_pos 9.f
#endif
#ifndef rconst_TransPoint_des_neg
#define rconst_TransPoint_des_neg 45
#endif
#ifndef rconst_smoothness_des_neg
#define rconst_smoothness_des_neg 10.f
#endif
#ifndef rconst_TransPoint_asc_neg
#define rconst_TransPoint_asc_neg 60
#endif
#ifndef rconst_smoothness_asc_neg
#define rconst_smoothness_asc_neg 9.f
#endif
    float rconst = 0;
    float rconst_TransPoint_des, rconst_smoothness_des, rconst_TransPoint_asc, rconst_smoothness_asc;
    if (Polarity > 0) {
        rconst_TransPoint_des = rconst_TransPoint_des_pos;
        rconst_smoothness_des = rconst_smoothness_des_pos;
        rconst_TransPoint_asc = rconst_TransPoint_asc_pos;
        rconst_smoothness_asc = rconst_smoothness_asc_pos;
    } else {
        rconst_TransPoint_des = rconst_TransPoint_des_neg;
        rconst_smoothness_des = rconst_smoothness_des_neg;
        rconst_TransPoint_asc = rconst_TransPoint_asc_neg;
        rconst_smoothness_asc = rconst_smoothness_asc_neg;
    }

    if (SolarPhase == 1) {
        rconst = SmoothTransition(MaxValueOf_rconst, 1, rconst_TransPoint_des, rconst_smoothness_des, tilt);
    } else {
        rconst = SmoothTransition(MaxValueOf_rconst, 1, rconst_TransPoint_asc, rconst_smoothness_asc, tilt);
    }
    return rconst;
}

__device__ float3 Diffusion_Tensor_In_HMF_Frame(const unsigned int InitZone, const signed int HZone, const float r,
                                                const float theta, const float beta, const float P,
                                                const float GaussRndNumber, float3 &dK_dr) {
    /*Authors: 2022 Stefano */
    /* * description: evaluate the diffusion tensor in the HMF frame, i.e. Kparallel & Kperpendicular.
        \param HZone   Zone in the Heliosphere
        \param r      solar distance
        \param theta  solar colatitude
        \param beta   v/c
        \param P      Particle rigidity
        \param GaussRndNumber Random number with normal distribution
        \return x Kparallel
                y Kperp_1
                z Kperp_2
    */
    float3 Ktensor;
    // HeliosphereZoneProperties_t ThisZone=LIM[HZone+InitZone];

    const int high_activity = Heliosphere.IsHighActivityPeriod[InitZone] ? 0 : 1;
    const float k0_paral = LIM[HZone + InitZone].k0_paral[high_activity];
    const float k0_perp = LIM[HZone + InitZone].k0_perp[high_activity];
    const float GaussVar = LIM[HZone + InitZone].GaussVar[high_activity];
    const float g_low = LIM[HZone + InitZone].g_low;
    const float rconst = LIM[HZone + InitZone].rconst;


    // Kpar = k0 * beta/3 * (P/1GV + glow)*( Rconst+r/1AU) with k0 gaussian distributed
    dK_dr.x = (k0_paral + GaussRndNumber * GaussVar * k0_paral) * beta / 3.f * (P + g_low);
    Ktensor.x = dK_dr.x * (rconst + r);

#ifndef rho_1
#define rho_1 0.065f // Kpar/Kperp (ex Kp0)
#endif
#ifndef PolarEnhanc
#define PolarEnhanc 2 // polar enhancement in polar region
#endif

    // Kperp1 = rho_1(theta)* k0 * beta/3 * (P/1GV + glow)*( Rconst+r/1AU)
    dK_dr.y = rho_1 * k0_perp * beta / 3.f * (P + g_low) * (fabsf(cosf(theta)) > CosPolarZone ? PolarEnhanc : 1.f);
    Ktensor.y = dK_dr.y * (rconst + r);

    // Kperp2 = rho_2 * k0 * beta/3 * (P/1GV + glow)*( Rconst+r/1AU) with rho_2=rho_1
    dK_dr.z = rho_1 * k0_perp * beta / 3.f * (P + g_low);
    Ktensor.z = dK_dr.z * (rconst + r);

    return Ktensor;
}

__device__ float Diffusion_Coeff_heliosheat(const unsigned int HZone, const float r, const float th, const float phi,
                                            const float beta, const float P, float &dK_dr) {
    /*Authors: 2022 Stefano */
    /* * description: evaluate the diffusion tensor in the HMF frame, i.e. Kparallel & Kperpendicular.
        \param HZone   Zone in the Heliosphere
        \param r      solar distance
        \param beta   v/c
        \param P      Particle rigidity
        \return x diffusion coeff
    */
    dK_dr = 0.;
    // if around 5 AU from Heliopause, apply diffusion barrier
    const float RhpDirection = Boundary(th, phi, Heliosphere.RadBoundary_effe[HZone].Rhp_nose,
                                        Heliosphere.RadBoundary_effe[HZone].Rhp_tail);
#ifndef HPB_SupK
#define HPB_SupK 50 // suppressive factor at barrier
#endif
#ifndef HP_width
#define HP_width 2 // amplitude in AU of suppressive factor at barrier
#endif
#ifndef HP_SupSmooth
#define HP_SupSmooth 3e-2 // smoothness of suppressive factor at barrier
#endif
    if (r > RhpDirection - 5) {
        return HS[HZone].k0 * beta * P * SmoothTransition(1, 1. / HPB_SupK, RhpDirection - HP_width / 2.f,
                                                          HP_SupSmooth, r);
    }
    return HS[HZone].k0 * beta * P;
}
