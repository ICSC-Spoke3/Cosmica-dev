#include "hip/hip_runtime.h"
#include <cstdio>
#include <hiprand/hiprand_kernel.h>  // CUDA random number device library
#include <hip/hip_runtime.h>
#include "HeliosphericPropagation.cuh"
#include "VariableStructure.cuh"
#include "HelModVariableStructure.cuh"
#include "HeliosphereModel.cuh"
#include "SDECoeffs.cuh"
#include "GenComputation.cuh"
#include "Histogram.cuh"

__global__ void HeliosphericProp(const unsigned int Npart_PerKernel, const float Min_dt, float Max_dt,
                                 const float TimeOut,
                                 ThreadQuasiParticles_t QuasiParts_out, const ThreadIndexes_t indexes,
                                 const HeliosphereZoneProperties_t *__restrict__ LIM,
                                 hiprandStatePhilox4_32_10_t *const CudaState,
                                 float *RMaxs) {
    const unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= Npart_PerKernel) return;

    extern __shared__ float smem[];

    hiprandStatePhilox4_32_10_t randState = CudaState[id];

    auto qp = QuasiParts_out.get(id);
    auto index = indexes.get(id);
    index.update(qp);


    while (index.radial >= 0 && qp.t_fly <= TimeOut) {
        const auto [rand_x, rand_y, rand_z, rand_w] = hiprand_normal4(&randState);

        auto KSym = DiffusionTensor_symmetric(index, qp, Heliosphere.Isotopes[index.particle], rand_w, LIM);

        int res = 0;
        const auto [rr, tr, tt, pr, pt, pp] = SquareRoot_DiffusionTerm(index, qp, KSym, &res);

        if (res > 0) {
            // SDE diffusion matrix is not positive definite; in this case propagation should be stopped and a new event generated
            // placing the energy below zero ensure that this event is ignored in the after-part of the analysis
            qp.R = -1;
            break; //exit the while cycle
        }

        const auto [adv_r, adv_th, adv_phi] = AdvectiveTerm(index, qp, KSym, Heliosphere.Isotopes[index.particle], LIM);

        const float en_loss = EnergyLoss(index, qp, LIM);

        const float dt = fmaxf(Min_dt, fminf(fminf(Max_dt,
                                                   Min_dt * (rr * rr) / (adv_r * adv_r)),
                                             Min_dt * (tr + tt) * (tr + tt) / (adv_th * adv_th)));

        if (const float update_r = qp.r + adv_r * dt + rand_x * rr * sqrtf(dt); update_r >= r_mirror) {
            qp.r = update_r;
            qp.th += adv_th * dt + (rand_x * tr + rand_y * tt) * sqrtf(dt);
            qp.phi += adv_phi * dt + (rand_x * pr + rand_y * pt + rand_z * pp) * sqrtf(dt);
            qp.R += en_loss * dt;
            qp.t_fly += dt;
        }

        qp.normalize_angles();

        index.update(qp);
    }

    QuasiParts_out.r[id] = qp.r;
    QuasiParts_out.th[id] = qp.th;
    QuasiParts_out.phi[id] = qp.phi;
    smem[threadIdx.x] = QuasiParts_out.R[id] = qp.R;
    QuasiParts_out.t_fly[id] = qp.t_fly;

    BlockMax(smem, RMaxs);
}
