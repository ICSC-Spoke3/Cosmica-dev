#include "hip/hip_runtime.h"
#include <cstdio>
#include <hiprand/hiprand_kernel.h>  // CUDA random number device library
#include <hip/hip_runtime.h>
#include "HeliosphericPropagation.cuh"
#include "VariableStructure.cuh"
#include "HelModVariableStructure.cuh"
#include "SDECoeffs.cuh"
#include "Histogram.cuh"

__global__ void HeliosphericProp(const unsigned Npart_PerKernel, ThreadQuasiParticles_t QuasiParts_out,
                                 const ThreadIndexes_t indexes, const SimulationParametrization_t params,
                                 hiprandStatePhilox4_32_10_t *const CudaState, float *RMaxs) {
    const unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= Npart_PerKernel) return;

    extern __shared__ float smem[];

    hiprandStatePhilox4_32_10_t randState = CudaState[id];

    auto qp = QuasiParts_out.get(id);
    auto index = indexes.get(id);
    index.update(qp);


    while (index.radial >= 0 && qp.t_fly <= Constants.Timeout) {
        const auto [rand_x, rand_y, rand_z, rand_w] = hiprand_normal4(&randState);

        auto KSym = DiffusionTensor_symmetric(index, qp, Constants.Isotopes[index.isotope], rand_w, params);

        int res = 0;
        const auto [rr, tr, tt, pr, pt, pp] = SquareRoot_DiffusionTerm(index, qp, KSym, &res);

        if (res > 0) {
            // SDE diffusion matrix is not positive definite; in this case propagation should be stopped and a new event generated
            // placing the energy below zero ensure that this event is ignored in the after-part of the analysis
            qp.R = -1;
            break; //exit the while cycle
        }

        const auto [adv_r, adv_th, adv_phi] = AdvectiveTerm(index, qp, KSym, Constants.Isotopes[index.isotope]);

        const float en_loss = EnergyLoss(index, qp);

        const float dt = fmaxf(Constants.Min_dt, fminf(
                                   fminf(Constants.Max_dt, Constants.Min_dt * (rr * rr) / (adv_r * adv_r)),
                                   Constants.Min_dt * (tr + tt) * (tr + tt) / (adv_th * adv_th)));

        if (const float update_r = qp.r + adv_r * dt + rand_x * rr * sqrtf(dt); update_r >= r_mirror) {
            qp.r = update_r;
            qp.th += adv_th * dt + (rand_x * tr + rand_y * tt) * sqrtf(dt);
            qp.phi += adv_phi * dt + (rand_x * pr + rand_y * pt + rand_z * pp) * sqrtf(dt);
            qp.R += en_loss * dt;
            qp.t_fly += dt;
        }

        qp.normalize_angles();

        index.update(qp);
    }

    QuasiParts_out.r[id] = qp.r;
    QuasiParts_out.th[id] = qp.th;
    QuasiParts_out.phi[id] = qp.phi;
    smem[threadIdx.x] = QuasiParts_out.R[id] = qp.R;
    QuasiParts_out.t_fly[id] = qp.t_fly;

    BlockMax(smem, RMaxs);
}
