
#include <hip/hip_runtime.h>
// .. credit to Mark Harris (https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf)

// Use of the tamplate to unroll the loop at compile time (kernel_6 optimization)
/* template <unsigned int blockSize>
__device__ void WarpMax(volatile float* sdata, unsigned int tid) {
    if (blockSize >= 64 && sdata[tid] < sdata[tid + 32]) sdata[tid] = sdata[tid + 32];
    if (blockSize >= 32 && sdata[tid] < sdata[tid + 16]) sdata[tid] = sdata[tid + 16];
    if (blockSize >= 16 && sdata[tid] < sdata[tid + 8])  sdata[tid] = sdata[tid + 8];
    if (blockSize >= 8  && sdata[tid] < sdata[tid + 4])  sdata[tid] = sdata[tid + 4];
    if (blockSize >= 4  && sdata[tid] < sdata[tid + 2])  sdata[tid] = sdata[tid + 2];
    if (blockSize >= 2  && sdata[tid] < sdata[tid + 1])  sdata[tid] = sdata[tid + 1];
}

template <unsigned int blockSize>
__device__ void BlockMax(float* sdata, float* outdata) {
    
    // thread index taking into account the shift imposed by the rigidity positions in shared memory array
    unsigned int tid = threadIdx.x + 3*blockSize;

    // first max search steps with sub-array larger than warp dimension (unrolled loop of max search)
    if (blockSize >= 512) {
        if (tid < 256 && sdata[tid] < sdata[tid + 256]) sdata[tid] = sdata[tid + 256];
            __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128 && sdata[tid] < sdata[tid + 128]) sdata[tid] = sdata[tid + 128];
            __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64 && sdata[tid] < sdata[tid + 64]) sdata[tid] = sdata[tid + 64];
            __syncthreads();
    }

    // warp reduction
    if (tid < 32) WarpMax(sdata, tid);

    // write result for this block to global mem
    if (tid == 0) outdata = sdata[0];
}

template <unsigned int blockSize>
__global__ void GridMax(float* indata, float* outdata) {
    // shared memory allocation and filling
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize + threadIdx.x;

    sdata[tid] = indata[i];
    __syncthreads();

    // first max search steps with sub-array larger than warp dimension (unrolled loop of max search)
    if (blockSize >= 512) {
        if (tid < 256 && sdata[tid] < sdata[tid + 256]) sdata[tid] = sdata[tid + 256];
            __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128 && sdata[tid] < sdata[tid + 128]) sdata[tid] = sdata[tid + 128];
            __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64 && sdata[tid] < sdata[tid + 64]) sdata[tid] = sdata[tid + 64];
            __syncthreads();
    }

    // warp reduction
    if (tid < 32) WarpMax(sdata, tid);

    // write result for this block to global mem
    if (tid == 0) outdata = sdata[0];
} */

// Unroll the last steps when reduction dimension < warp dimension
__device__ void WarpMax(volatile float *sdata, const unsigned int tid) {
    if (sdata[tid] < sdata[tid + 32]) sdata[tid] = sdata[tid + 32];
    if (sdata[tid] < sdata[tid + 16]) sdata[tid] = sdata[tid + 16];
    if (sdata[tid] < sdata[tid + 8]) sdata[tid] = sdata[tid + 8];
    if (sdata[tid] < sdata[tid + 4]) sdata[tid] = sdata[tid + 4];
    if (sdata[tid] < sdata[tid + 2]) sdata[tid] = sdata[tid + 2];
    if (sdata[tid] < sdata[tid + 1]) sdata[tid] = sdata[tid + 1];
}

__device__ void BlockMax(float *sdata, float *outdata) {
    // thread index taking into account the shift imposed by the rigidity positions in shared memory array
    const unsigned int sdata_id = threadIdx.x + 3 * blockDim.x;

    // first max search steps with sub-array larger than warp dimension
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (threadIdx.x < s && sdata[sdata_id] < sdata[sdata_id + s]) sdata[sdata_id] = sdata[sdata_id + s];
        __syncthreads();
    }

    // warp reduction
    if (threadIdx.x < 32) WarpMax(sdata, sdata_id);

    // write result for this block to global mem
    if (threadIdx.x == 0) outdata[blockIdx.x] = sdata[sdata_id];
}

__global__ void GridMax(const int Nmax, const float *indata, float *outdata) {
    // shared memory allocation and filling
    extern __shared__ float sdata[];

    // !!!This is useful only launched recursively on different blocks
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < Nmax) sdata[threadIdx.x] = indata[id];
    else if (id >= Nmax) sdata[threadIdx.x] = 0;

    __syncthreads();

    // first max search steps with sub-array larger than warp dimension
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (threadIdx.x < s && sdata[threadIdx.x] < sdata[threadIdx.x + s]) sdata[threadIdx.x] = sdata[threadIdx.x + s];
        __syncthreads();
    }

    // warp reduction
    if (threadIdx.x < 32) WarpMax(sdata, threadIdx.x);

    // write result for this block to global mem
    if (threadIdx.x == 0) outdata[blockIdx.x] = sdata[0];
}

__global__ void Rhistogram_atomic(const float *R_in, const float LogBin0_lowEdge, const float DeltaLogR, const int Nbin,
                                  const unsigned int Npart, float *R_out) {
    extern __shared__ unsigned int smem[];

    const unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int block_shift = blockIdx.x * Nbin;

    // initialize the shared memory empty histogram
    if (threadIdx.x < Nbin) smem[threadIdx.x] = 0;

    __syncthreads();

    if (id < Npart) {
        if (log10f(R_in[id]) > LogBin0_lowEdge) {
            // evalaute the bin where put event and add atomically
            const int dest_bin = static_cast<int>(floorf((log10f(R_in[id]) - LogBin0_lowEdge) / DeltaLogR));
            atomicAdd(&smem[dest_bin], 1);
        }
    }

    // write partial histogram to global memory
    if (threadIdx.x < Nbin) R_out[threadIdx.x + block_shift] = static_cast<float>(smem[threadIdx.x]);
}

// Unroll the last steps when reduction dimension < warp dimension
__device__ void WarpSum(volatile int *sdata, const unsigned int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__global__ void TotalHisto(const float *indata, const unsigned int Nbins, const unsigned int Nblocks, float *outdata) {
    // shared memory allocation and filling
    extern __shared__ int shist[];

    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int bin_id = 2 * threadIdx.x * Nbins + blockIdx.x;

    // First histogram couple merge during shared memory allocation
    // Each block perform one rigidity bin reduction
    if (id < Nbins * Nblocks) shist[threadIdx.x] = static_cast<int>(indata[bin_id] + indata[bin_id + Nbins]);

    __syncthreads();

    // first max search steps with sub-array larger than warp dimension
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (threadIdx.x < s) shist[threadIdx.x] += shist[threadIdx.x + s];
        __syncthreads();
    }

    // warp reduction
    if (threadIdx.x < 32) WarpSum(shist, threadIdx.x);

    // write result for this block to global mem
    if (threadIdx.x == 0) outdata[blockIdx.x] = static_cast<float>(shist[0]);
}

////////////////////////////////////////////////////////////////
//..... histogram handling .....................................
//
////////////////////////////////////////////////////////////////

__global__ void histogram_atomic(const float *in, const float LogBin0_lowEdge, const float DeltaLogT, const int Nbin,
                                 const unsigned long Npart, float *out, int *Nfailed) {
    // NOTE: not using shared memory

    const unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    for (unsigned int it = threadIdx.x; it < Nbin; it += blockDim.x) {
        out[blockIdx.x * Nbin + it] = 0;
    }

    __syncthreads();

    if (id < Npart) {
        if (log10f(in[id]) > LogBin0_lowEdge) {
            const int DestBin = static_cast<int>(floorf((log10f(in[id]) - LogBin0_lowEdge) / DeltaLogT));
            // evalaute the bin where put event
            atomicAdd(&out[blockIdx.x * Nbin + DestBin], 1); // exp(alphapath[id])
        } else {
            atomicAdd(Nfailed, 1);
            // nota per futuro. le particelle uccise hanno valori diversi negativi in base all'evento che li ha uccisi,
            //                  quindi se Nfailed diventasse una struct con il tipo di errore, si potrebbe fare una
            //                  statistica dettagliata dell'errore.
        }
    }
}

__global__ void histogram_accum(const float *in, const int Nbins, const int NFraz, float *out) {
    // NOTE: not using shared memory

    const unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id >= Nbins) { return; } //out of range

    float total = 0.;

    for (int ithb = 0; ithb < NFraz; ithb++) {
        total += in[id + ithb * Nbins];
    }

    out[id] = total;
}
