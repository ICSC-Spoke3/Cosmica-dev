#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>          // Supplies FILE, stdin, stdout, stderr, and the fprint() family of functions
#include <hip/hip_runtime.h>   // Device code management by providing implicit initialization, context management, and module management


#include "GPUManage.cuh"
#include "VariableStructure.cuh"
#include "GenComputation.cuh"

/**
 * @brief HandleError
 * @param err hipError_t error
 * @param file the file where the error occurred
 * @param line the line where the error occurred
 */
void HandleError(const hipError_t err, const char *file, const int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

/**
 * @brief hash function to convert a string into a hash
 * @param str the string to be hashed
 * @return the hash of the string
 */
constexpr unsigned long hash(const std::string_view &str) {
    unsigned long hash = 0;
    for (const auto &e: str) hash = hash * 131 + e;
    return hash;
}

/**
 * @brief operator""_ to convert a string into a hash
 * @param str the string to be hashed
 * @param len the length of the string
 * @return the hash of the string
 */
consteval unsigned long operator""_(const char *str, const size_t len) {
    return hash(std::string_view(str, len));
}

/**
 * @brief Define the best value of NWarpPerBlock for a given GPU (name)
 * @param name the name of the GPU
 * @param verbose the verbosity of the output
 * @return the best warp per block
 */
int BestWarpPerBlock(char name[], const int verbose) {
    int BestWarpPerBlock = 8;

    switch (hash(name)) {
        case "NVIDIA A30"_:
        case "NVIDIA A40"_:
            BestWarpPerBlock = 2;
            break;
        case "NVIDIA A100"_:
            BestWarpPerBlock = 16;
            break;
        default:
            std::cerr << "WARNING: best value not known, used default warp per block = 8 for " << name << std::endl;
    }

    if (verbose) {
        printf("----- Simulation infos -----\n");
        printf("-- For board %s we execute the code using NWarpPerBlock=%d\n", name, BestWarpPerBlock);
    }

    return BestWarpPerBlock;
}

/**
 * @brief Round the number of particle to be simulated based on the GPU capability, returning a struct with threads
 * and blocks count together with shared memory bytes.
 * The calculations are done based on the cuda occupancy calculator output to maximize the usage of the GPUs.
 * @param NPart the number of particles
 * @param GPUprop the GPU properties
 * @param verbose the verbosity of the output
 * @param WpB the warp per block
 * @return the launch parameters
 */
LaunchParam_t RoundNpart(const unsigned NPart, hipDeviceProp_t GPUprop, const bool verbose, const int WpB) {
    LaunchParam_t launch_param;

    // Computation of the number of blocks, warp per blocks, threads per block and shared memory bits
    int WarpPerBlock = WpB <= 0 ? BestWarpPerBlock(GPUprop.name, verbose) : WpB;
    launch_param.threads = WarpPerBlock * GPUprop.warpSize;
    launch_param.blocks = ceil_int_div(NPart, launch_param.threads);
    // Use a minimum of 2 blocks per Single Multiprocessor (cuda prescription)
    if (launch_param.blocks < 2) launch_param.blocks = 2;

    if (launch_param.threads > static_cast<unsigned>(GPUprop.maxThreadsPerBlock) ||
        launch_param.blocks > static_cast<unsigned>(GPUprop.maxGridSize[0])) {
        fprintf(stderr, "------- propagation Kernel -----------------\n");
        fprintf(stderr, "ERROR:: Number of Threads per block or number of blocks not allowed for this device\n");
        fprintf(stderr, "        Number of Threads per Block setted %d - max allowed %d\n", launch_param.threads,
                GPUprop.maxThreadsPerBlock);
        fprintf(stderr, "        Number of Blocks setted %d - max allowed %d\n", launch_param.blocks,
                GPUprop.maxGridSize[0]);
        exit(EXIT_FAILURE);
    }

#define EXPERIMENTAL_GRID
#ifdef EXPERIMENTAL_GRID
    // int gridSize, minGridSize, blockSize = 32;
    // int maxActiveBlocks = 0;
    // hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, HeliosphericProp, blockSize, 0);
    // gridSize = GPUprop.multiProcessorCount * maxActiveBlocks;
    // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, HeliosphericProp, 0, 65536 / 86);
    // blockSize = (blockSize + GPUprop.warpSize - 1) / GPUprop.warpSize * GPUprop.warpSize;
    // gridSize = (NPart + blockSize - 1) / blockSize;

    // launch_param.threads = (768 + GPUprop.warpSize - 1) / GPUprop.warpSize * GPUprop.warpSize;
    launch_param.threads = 48;
    launch_param.blocks = (NPart + launch_param.threads - 1) / launch_param.threads;
#endif

    if (verbose) {
        printf("------- propagation Kernel -----------------\n");
        printf("-- Number of particle which will be simulated: %d\n", NPart);
        printf("-- Number of Warp in a Block       : %d \n", WarpPerBlock);
        printf("-- Number of blocks                : %d \n", launch_param.blocks);
        printf("-- Number of threadsPerBlock       : %d \n", launch_param.threads);
        printf("-- \n\n");
    }

    return launch_param;
}

/**
 * @brief Retrieve the GPU properties (info) for the selected GPU and print a summary of useful infos for debugging,
 * with the verbose option.
 * @param N_GPU_count the number of GPU
 * @param verbose the verbosity of the output
 * @return the GPU properties
 */
hipDeviceProp_t *DeviceInfo(const int N_GPU_count, const bool verbose) {
    const auto infos = new hipDeviceProp_t[N_GPU_count];

    for (int i = 0; i < N_GPU_count; i++)
        hipGetDeviceProperties(&infos[i], i);

    if (verbose) {
        printf("----- GPU infos -----\n");
        printf("There are %d CUDA enabled devices \n", N_GPU_count);

        for (int i = 0; i < N_GPU_count; i++) {
            printf("--   --- General Information for device %d ---\n", i);
            printf("-- Name:  %s\n", infos[i].name);
            printf("-- Compute capability:  %d.%d\n", infos[i].major, infos[i].minor);
            printf("-- Clock rate:  %d\n", infos[i].clockRate);
            printf("-- Device copy overlap:  ");
            if (infos[i].deviceOverlap)
                printf("Enabled\n");
            else
                printf("Disabled\n");
            printf("-- Kernel execution timeout :  ");
            if (infos[i].kernelExecTimeoutEnabled)
                printf("Enabled\n");
            else
                printf("Disabled\n");

            printf("--    --- Memory Information for device %d ---\n", i);
            printf("-- Total global mem:  %ld\n", infos[i].totalGlobalMem);
            printf("-- Total constant Mem:  %ld\n", infos[i].totalConstMem);
            printf("-- Max mem pitch:  %ld\n", infos[i].memPitch);
            printf("-- Texture Alignment:  %ld\n", infos[i].textureAlignment);

            printf("--    --- MP Information for device %d ---\n", i);
            printf("-- Multiprocessor count:  %d\n",
                   infos[i].multiProcessorCount);
            printf("-- Shared mem per mp:  %ld\n", infos[i].sharedMemPerBlock);
            printf("-- Registers per mp:  %d\n", infos[i].regsPerBlock);
            printf("-- Threads in warp:  %d\n", infos[i].warpSize);
            printf("-- Max threads per block:  %d\n",
                   infos[i].maxThreadsPerBlock);
            printf("-- Max thread dimensions:  (%d, %d, %d)\n",
                   infos[i].maxThreadsDim[0], infos[i].maxThreadsDim[1],
                   infos[i].maxThreadsDim[2]);
            printf("-- Max grid dimensions:  (%d, %d, %d)\n",
                   infos[i].maxGridSize[0], infos[i].maxGridSize[1],
                   infos[i].maxGridSize[2]);
            printf("-- \n\n");
        }
    }

    return infos;
}
