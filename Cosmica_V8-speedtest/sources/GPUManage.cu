#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>          // Supplies FILE, stdin, stdout, stderr, and the fprint() family of functions
#include <hip/hip_runtime.h>   // Device code management by providing implicit initialization, context management, and module management



#include "GPUManage.cuh"
#include "VariableStructure.cuh"
#include "GenComputation.cuh"

/**
 * @brief HandleError
 * @param err hipError_t error
 * @param file the file where the error occurred
 * @param line the line where the error occurred
 */
void HandleError(const hipError_t err, const char *file, const int line) {
    if (err != hipSuccess) {
        spdlog::critical("{} in {} at line {}", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

/**
 * @brief hash function to convert a string into a hash
 * @param str the string to be hashed
 * @return the hash of the string
 */
constexpr unsigned long hash(const std::string_view &str) {
    unsigned long hash = 0;
    for (const auto &e: str) hash = hash * 131 + e;
    return hash;
}

/**
 * @brief operator""_ to convert a string into a hash
 * @param str the string to be hashed
 * @param len the length of the string
 * @return the hash of the string
 */
consteval unsigned long operator""_(const char *str, const size_t len) {
    return hash(std::string_view(str, len));
}

/**
 * @brief Print the available GPUs
 * @return the number of GPUs
 */
int AvailableGPUs() {
    int NGPUs;
    HANDLE_ERROR(hipGetDeviceCount(&NGPUs));
    return NGPUs;
}


/**
 * @brief Define the best value of NWarpPerBlock for a given GPU (name)
 * @param name the name of the GPU
 * @param verbose the verbosity of the output
 * @return the best warp per block
 */
// TODO: refactor to be BestThreadsPerBlock
int BestWarpPerBlock(char name[], const int verbose) {
    int BestWarpPerBlock = 8;

    switch (hash(name)) {
        case "NVIDIA A30"_:
        case "NVIDIA A40"_:
            BestWarpPerBlock = 2;
            break;
        case "NVIDIA A100"_:
            BestWarpPerBlock = 16;
            break;
        default:
            std::cerr << "WARNING: best value not known, used default warp per block = 8 for " << name << std::endl;
    }

    if (verbose) {
        printf("----- Simulation infos -----\n");
        printf("-- For board %s we execute the code using NWarpPerBlock=%d\n", name, BestWarpPerBlock);
    }

    return BestWarpPerBlock;
}

/**
 * @brief Round the number of particle to be simulated based on the GPU capability, returning a struct with threads
 * and blocks count together with shared memory bytes.
 * The calculations are done based on the cuda occupancy calculator output to maximize the usage of the GPUs.
 * @param NPart the number of particles
 * @param GPUprop the GPU properties
 * @param verbose the verbosity of the output
 * @param WpB the warp per block
 * @return the launch parameters
 */
LaunchParam_t RoundNpart(const unsigned NPart, hipDeviceProp_t GPUprop, const bool verbose, const int WpB) {
    LaunchParam_t launch_param;

    // Computation of the number of blocks, warp per blocks, threads per block and shared memory bits
    int WarpPerBlock = WpB <= 0 ? BestWarpPerBlock(GPUprop.name, verbose) : WpB;
    launch_param.threads = WarpPerBlock * GPUprop.warpSize;
    launch_param.blocks = ceil_int_div(NPart, launch_param.threads);
    // Use a minimum of 2 blocks per Single Multiprocessor (cuda prescription)
    if (launch_param.blocks < 2) launch_param.blocks = 2;

    if (launch_param.threads > static_cast<unsigned>(GPUprop.maxThreadsPerBlock)) {
        spdlog::critical("Error while configuring the Propagation Kernel");
        spdlog::critical("Too many threads per block: {} (max allowed {})", launch_param.threads,
                         GPUprop.maxThreadsPerBlock);
        exit(EXIT_FAILURE);
    }
    if (launch_param.blocks > static_cast<unsigned>(GPUprop.maxGridSize[0])) {
        spdlog::critical("Error while configuring the Propagation Kernel");
        spdlog::critical("Too many blocks per grid: {} (max allowed {})", launch_param.blocks, GPUprop.maxGridSize[0]);
        exit(EXIT_FAILURE);
    }

#define EXPERIMENTAL_GRID
#ifdef EXPERIMENTAL_GRID
    // int gridSize, minGridSize, blockSize = 32;
    // int maxActiveBlocks = 0;
    // hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, HeliosphericProp, blockSize, 0);
    // gridSize = GPUprop.multiProcessorCount * maxActiveBlocks;
    // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, HeliosphericProp, 0, 65536 / 86);
    // blockSize = (blockSize + GPUprop.warpSize - 1) / GPUprop.warpSize * GPUprop.warpSize;
    // gridSize = (NPart + blockSize - 1) / blockSize;

    // launch_param.threads = (768 + GPUprop.warpSize - 1) / GPUprop.warpSize * GPUprop.warpSize;
    launch_param.threads = 48;
    launch_param.blocks = (NPart + launch_param.threads - 1) / launch_param.threads;
#endif

    spdlog::info("Propagation Kernel Configuration:");
    spdlog::info("* Number of particles      : {}", NPart);
    spdlog::info("* Number of Warp in a Block: {}", WarpPerBlock);
    spdlog::info("* Number of blocks         : {}", launch_param.blocks);
    spdlog::info("* Number of threadsPerBlock: {}", launch_param.threads);

    return launch_param;
}

/**
 * @brief Retrieve the GPU properties (info) for the selected GPU and print a summary of useful infos for debugging,
 * with the verbose option.
 * @param N_GPU_count the number of GPU
 * @return the GPU properties
 */
hipDeviceProp_t *DeviceInfo(const int N_GPU_count) {
    const auto infos = new hipDeviceProp_t[N_GPU_count];

    for (int i = 0; i < N_GPU_count; i++)
        hipGetDeviceProperties(&infos[i], i);

    spdlog::debug("GPU info:");

    for (int i = 0; i < N_GPU_count; i++) {
        spdlog::debug("* General Information for device {}:", i);
        spdlog::debug("  - Name:  {}", infos[i].name);
        spdlog::debug("  - Compute capability:  {}.{}", infos[i].major, infos[i].minor);
        spdlog::debug("  - Clock rate:  {}", infos[i].clockRate);
        spdlog::debug("  - Device copy overlap:  {}", infos[i].deviceOverlap ? "Enabled" : "Disabled");
        spdlog::debug("  - Kernel execution timeout :  {}", infos[i].kernelExecTimeoutEnabled ? "Enabled" : "Disabled");

        spdlog::debug("* Memory Information for device {}:", i);
        spdlog::debug("  - Total global mem:  {}", infos[i].totalGlobalMem);
        spdlog::debug("  - Total constant Mem:  {}", infos[i].totalConstMem);
        spdlog::debug("  - Max mem pitch:  {}", infos[i].memPitch);
        spdlog::debug("  - Texture Alignment:  {}", infos[i].textureAlignment);

        spdlog::debug("* MP Information for device {}:", i);
        spdlog::debug("  - Multiprocessor count:  {}", infos[i].multiProcessorCount);
        spdlog::debug("  - Shared mem per mp:  {}", infos[i].sharedMemPerBlock);
        spdlog::debug("  - Registers per mp:  {}", infos[i].regsPerBlock);
        spdlog::debug("  - Threads in warp:  {}", infos[i].warpSize);
        spdlog::debug("  - Max threads per block:  {}", infos[i].maxThreadsPerBlock);
        spdlog::debug("  - Max thread dimensions:  ({}, {}, {})", infos[i].maxThreadsDim[0], infos[i].maxThreadsDim[1],
                     infos[i].maxThreadsDim[2]);
        spdlog::debug("  - Max grid dimensions:  ({}, {}, {})", infos[i].maxGridSize[0], infos[i].maxGridSize[1],
                     infos[i].maxGridSize[2]);
    }

    return infos;
}
