#include "hip/hip_runtime.h"
#define MAINCU

// .. standard C
#include <stdio.h>          // Supplies FILE, stdin, stdout, stderr, and the fprint() family of functions
#include <stdlib.h>         // Supplies malloc(), calloc(), and realloc()
#include <unistd.h>         // Supplies EXIT_FAILURE, EXIT_SUCCESS
#include <libgen.h>         // Supplies the basename() function 
#include <errno.h>          // Defines the external errno variable and all the values it can take on
#include <string.h>         // Supplies memcpy(), memset(), and the strlen() family of functions
#include <getopt.h>         // Supplies external optarg, opterr, optind, and getopt() function
#include <sys/types.h>      // Typedef shortcuts like uint32_t and uint64_t
#include <sys/time.h>       // supplies time()

// .. multi-thread
#include <omp.h>

// math lib
#include <math.h>           // C math library
#include <limits.h>         // numerical C limits 
// .. CUDA specific
#include <hiprand.h>         // CUDA random number host library
#include <hiprand/hiprand_kernel.h>  // CUDA random number device library
#include <hip/hip_runtime.h>   // Device code management by providing implicit initialization, context management, and module management

// .. project specific
#include "VariableStructure.cuh"
#include "SDECoeffs.cuh"
#include "LoadConfiguration.cuh"
#include "HeliosphericPropagation.cuh"
#include "HeliosphereLocation.cuh"
#include "GenComputation.cuh"
#include "HistoComputation.cuh"
#include "GPUManage.cuh"
#include "Histogram.cuh"

// .. old HelMod code
#include "HelModVariableStructure.cuh"
#include "HelModLoadConfiguration.cuh"
#include "MagneticDrift.cuh"
#include "SolarWind.cuh"
#include "DiffusionModel.cuh"
#include "HeliosphereModel.cuh"

// Track the errors
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define ERR_NoOutputFile "ERROR: output file cannot be open, do you have writing permission?\n"

// Simulation iperparameters definition
#define WARPSIZE 32
#ifndef SetWarpPerBlock
  #define SetWarpPerBlock -1                                          // number of warp so be submitted -- modify this value to find the best performance
#endif
#define NPARTS 5000
#ifndef MAX_DT
  #define MAX_DT 50.                                        // max allowed value of time step
#endif
#ifndef MIN_DT
  #define MIN_DT 0.01                                       // min allowed value of time step
#endif
#define TIMEOUT std::numeric_limits<float>::infinity()
#define NPOS 10
#define RBINS 100

// Debugging variables
#define VERBOSE 1
#define VERBOSE_2 1
#define VERBOSE_LOAD 3
#define SINGLE_CPU 1
#define HELMOD_LOAD 1
#define INITSAVE 1
#define FINALSAVE 1
#define TRIVIAL 0
#define NVIDIA_HIST 0

// Datas variables
#define MaxCharinFileName   90

// -----------------------------------------------------------------
// ------------  Device Constant Variables declaration -------------
// -----------------------------------------------------------------
__constant__ SimulatedHeliosphere_t      Heliosphere;            // Heliosphere properties include Local Interplanetary medium parameters
__constant__ HeliosphereZoneProperties_t LIM[NMaxRegions];       // inner heliosphere
__constant__ HeliosheatProperties_t      HS[NMaxRegions];        // heliosheat
// __constant__ float dev_Npart;
// __constant__ float min_dt;
// __constant__ float max_dt;
// __constant__ float timeout;


// Main Code
int main(int argc, char* argv[]) {

    ////////////////////////////////////////////////////////////////
    //..... Print Start time  ...................................
    // This part is for debug and performances tests
    ////////////////////////////////////////////////////////////////
    if (VERBOSE)
    {
        // -- Save initial time of simulation
        time_t tim =time(NULL);
        struct tm *local = localtime(&tim);
        printf("\nSimulation started at: %s  \n",asctime(local));
    }
    ////////////////////////////////////////////////////////////////


    ////////////////////////////////////////////////////////////////
    //..... Initialize CPU threads   ...............................
    ////////////////////////////////////////////////////////////////

    //  Run as many CPU threads as there are CUDA devices
    //  each CPU thread controls a different device, processing its
    //  portion of the data.
    //  Initialize the global simulation parameter fron the input file
    //  Start execution time recording


    // Retrive GPUs infos and set the CPU multi threads
    int NGPUs;
    HANDLE_ERROR(hipGetDeviceCount(&NGPUs));   // Count the available GPUs
    
    if (NGPUs<1) {
        fprintf(stderr,"No CUDA capable devices were detected\n");
        exit(EXIT_FAILURE);
    }

    // Retrive the infos of alla the available GPUs and eventually print them
    hipDeviceProp_t* GPUs_profile = DeviceInfo(NGPUs, VERBOSE);

    omp_set_num_threads(NGPUs);                 // create as many CPU threads as there are CUDA devices
    unsigned int num_cpu_threads = omp_get_num_threads();  // numero totale di CPU-thread allocated

    printf("\n");
    printf("----- Global CPU infos -----\n" );
    printf("Number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("Number of host CPUs allocated:\t%d\n", num_cpu_threads);

    #if SINGLE_CPU
        omp_set_num_threads(1);   // setting 1 CPU thread for easier debugging
        NGPUs = 1;                // setting 1 GPU thread for easier debugging

        if (VERBOSE) {
            printf("WARNING: only 1 CPU managing only 1 GPU thread is instanziated, for easier debugging\n\n");
        }
    #endif

    // Allocate the intial positions and rigidities into which load simulation configuration values
    struct InitialPositions_t InitialPositions;
    float* InitialRigidities;

    // Allocate simulation global parameters
    int NInitPos = 0;
    int NParts = 0;
    int NInitRig = 0;
    float RelativeBinAmplitude = 0;
    struct SimParameters_t SimParameters;
    struct PartDescription_t pt;

    #if HELMOD_LOAD

        // NOTE: USING OLD STABLE 4_CoreCode_MultiGPU_MultiYear VERSION
        if (Load_Configuration_File(argc, argv, SimParameters, VERBOSE) != EXIT_SUCCESS) {
            printf("Error while loading simulation parameters\n");
            exit(EXIT_FAILURE);
        }

        // Initialize the needed parameters for the new cosmica code
        NInitPos = (int)SimParameters.NInitialPositions;
        NParts = (int)SimParameters.Npart;
        InitialPositions = LoadInitPos(NParts, VERBOSE);
        pt = SimParameters.IonToBeSimulated;

        ////////////////////////////////////////////////////////////////
        //..... Rescale Heliosphere to an effective one  ...............
        ////////////////////////////////////////////////////////////////

        for (int ipos=0; ipos<NInitPos; ipos++) {
            SimParameters.HeliosphereToBeSimulated.RadBoundary_effe[ipos] = SimParameters.HeliosphereToBeSimulated.RadBoundary_real[ipos];
            RescaleToEffectiveHeliosphere(SimParameters.HeliosphereToBeSimulated.RadBoundary_effe[ipos], SimParameters.InitialPosition[ipos]);
            
            if (VERBOSE_2){
                fprintf(stderr,"--- Zone %d \n", ipos);
                fprintf(stderr,"--- !! Effective Heliosphere --> effective boundaries: TS_nose=%f TS_tail=%f Rhp_nose=%f  Rhp_tail=%f \n", SimParameters.HeliosphereToBeSimulated.RadBoundary_effe[ipos].Rts_nose,SimParameters.HeliosphereToBeSimulated.RadBoundary_effe[ipos].Rts_tail,SimParameters.HeliosphereToBeSimulated.RadBoundary_effe[ipos].Rhp_nose,SimParameters.HeliosphereToBeSimulated.RadBoundary_effe[ipos].Rhp_tail);
                fprintf(stderr,"--- !! Effective Heliosphere --> new Source Position: r=%f th=%f phi=%f \n", SimParameters.InitialPosition[ipos].r, SimParameters.InitialPosition[ipos].th,SimParameters.InitialPosition[ipos].phi);
            }
        
            // Copy initial positions from SimParameters to the CPU InitialPositions_t
            InitialPositions.r[ipos] = SimParameters.InitialPosition[ipos].r;
            InitialPositions.th[ipos] = SimParameters.InitialPosition[ipos].th;
            InitialPositions.phi[ipos] = SimParameters.InitialPosition[ipos].phi;
        }

        NInitRig = (int)SimParameters.NT;
        InitialRigidities = LoadInitRigidities(NInitRig, VERBOSE);

        for (int iR=0; iR<NInitRig; iR++) {
            InitialRigidities[iR] = SimParameters.Tcentr[iR];
        }

        // relative (respect 1.) amplitude of Energy bin used as X axis in BoundaryDistribution  --> delta T = T*RelativeBinAmplitude
        RelativeBinAmplitude = SimParameters.RelativeBinAmplitude;
    
    // Load the global simulation parameters with new cosmica-GC method
    #else
        // Load the initial positions and particle number to simulate
        InitialPositions = LoadInitPos(NPOS, VERBOSE_2);
        NInitPos = NPOS;
        NParts = NPARTS;

        // Load rigidities to be simulated
        InitialRigidities = LoadInitRigidities(RBINS, VERBOSE_2);
        NInitRig = RBINS;

        // relative (respect 1.) amplitude of Energy bin used as X axis in BoundaryDistribution  --> delta T = T*RelativeBinAmplitude
        RelativeBinAmplitude = 0.00855;

    // Load the global simulation parameters with old HelMod method
    /*
    typedef struct SimParameters_t {                                                    // Place here all simulation variables
        char  output_file_name[struct_string_lengh]="SimTest";
        unsigned long      Npart=5000;                                  // number of event to be simulated
        unsigned char      NT;                                          // number of bins of energies to be simulated
        unsigned char      NInitialPositions=0;                         // number of initial positions -> this number represent also the number of Carrington rotation that                 
        float              *Tcentr;                                     // array of energies to be simulated
        vect3D_t           *InitialPosition;                            // initial position
        PartDescription_t  IonToBeSimulated;                            // Ion to be simulated
        MonteCarloResult_t *Results;                                    // output of the code
        float RelativeBinAmplitude = 0.00855 ;                          // relative (respect 1.) amplitude of Energy bin used as X axis in BoundaryDistribution  --> delta T = T*RelativeBinAmplitude
        SimulatedHeliosphere_t HeliosphereToBeSimulated;                // Heliosphere properties for the simulation
        HeliosphereZoneProperties_t prop_medium[NMaxRegions];           // PROPerties of the interplanetary MEDIUM - Heliospheric Parameters in each Heliospheric Zone
        HeliosheatProperties_t prop_Heliosheat[NMaxRegions];            // Properties of Heliosheat
    } SimParameters_t;
    */
    #endif

    // Allocation of the output results for all the rigidities
    struct MonteCarloResult_t* Results = (struct MonteCarloResult_t*)malloc(NInitRig*sizeof(MonteCarloResult_t));

    // .. Results saving files

    // Initial and final results files
    char file_trivial[8];
    #if TRIVIAL
        sprintf(file_trivial, "trivial_");
    #else
        sprintf(file_trivial, "");
    #endif

    char init_filename[20];
    sprintf(init_filename, "%sprop_in.txt", file_trivial);
    char final_filename[ReadingStringLenght];
    sprintf(final_filename, "%s_%sprop_out.txt", SimParameters.output_file_name, file_trivial);
    // Clean previous files
    if (remove(init_filename) != 0 || remove(final_filename) != 0) printf("Error deleting the old propagation files or it does not exist\n");
    else printf("Old propagation files deleted successfully\n");

    // Initial and final results files
    char histo_filename[20];
    sprintf(histo_filename, "%sR_histo.txt", file_trivial);
    // Clean previous files
    if (remove(histo_filename) != 0) printf("Error deleting the old histogram files or it does not exist\n");
    else printf("Old histogram files deleted successfully\n");
    printf( "-- \n\n" );


    ////////////////////////////////////////////////////////////////
    //..... Simulations initialization   ...........................
    ////////////////////////////////////////////////////////////////

    //  Start CPU pragma menaging its own portion of the data
    //  Optimation of the number of particles, threads, blocks and 
    //  shared memory with respect the GPU hardware


    // start cpu threads
    #pragma omp parallel
    {
        // Grep the CPU and GPU id and set them
        unsigned int cpu_thread_id = omp_get_thread_num();     // identificativo del CPU-thread
        int gpu_id = cpu_thread_id % NGPUs;              // seleziona la id della GPU da usare. "% num_gpus" allows more CPU threads than GPU devices
        HANDLE_ERROR(hipSetDevice(gpu_id));                   // seleziona la GPU

        if (VERBOSE) {
            printf( "----- Individual CPU infos -----\n" );
            printf("-- CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id, num_cpu_threads, gpu_id);
            printf( "-- \n\n" );
        }


        // Retrive information from the set GPU
        hipDeviceProp_t device_prop = GPUs_profile[gpu_id];

        // Rounding the number of particle and calculating threads, blocks and share memory to acheive the maximum usage of the GPUs
        LaunchParam_t prop_launch_param = RoundNpart(NParts, device_prop, VERBOSE, SetWarpPerBlock);

        ////////////////////////////////////////////////////////////////
        //..... capture the start time of GPU part .....................
        //      This part is for debug and performances tests
        ////////////////////////////////////////////////////////////////
        hipEvent_t     start,MemorySet,Randomstep, stop;
        hipEvent_t     Cycle_start,Cycle_step00,Cycle_step0,Cycle_step1,Cycle_step2, InitialSave, FinalSave;
        if (VERBOSE){
        HANDLE_ERROR( hipEventCreate( &start ) );
        HANDLE_ERROR( hipEventCreate( &MemorySet ) );
        HANDLE_ERROR( hipEventCreate( &Randomstep ) );
        HANDLE_ERROR( hipEventCreate( &stop ) );
        HANDLE_ERROR( hipEventRecord( start, 0 ) );
        }
        ////////////////////////////////////////////////////////////////


        ////////////////////////////////////////////////////////////////
        //..... GPU execution initialization   .........................
        ////////////////////////////////////////////////////////////////

        //  Set pseudo random number generator seeds
        //  Device memory allocation and threads starting positions


        // .. Initialize random generator
        hiprandStatePhilox4_32_10_t *dev_RndStates;
        HANDLE_ERROR(hipMalloc((void **)&dev_RndStates, prop_launch_param.Npart*sizeof(hiprandStatePhilox4_32_10_t)));
        unsigned long Rnd_seed=getpid()+time(NULL)+gpu_id;
        init_rdmgenerator<<<prop_launch_param.blocks, prop_launch_param.threads>>>(dev_RndStates, Rnd_seed);
        hipDeviceSynchronize();

        if (VERBOSE){
            //.. capture the time from GPU
            HANDLE_ERROR( hipEventRecord( Randomstep, 0 ) );
            HANDLE_ERROR( hipEventSynchronize( Randomstep ) );
            if (VERBOSE>=VERBOSE_med){
              fprintf(stdout,"--- Random Generator Seed: %lu \n",Rnd_seed);
            }
        }

        // .. copy heliosphere parameters to Device Constant Memory
        hipMemcpyToSymbol(HIP_SYMBOL(Heliosphere), &SimParameters.HeliosphereToBeSimulated, sizeof(SimulatedHeliosphere_t));
        hipMemcpyToSymbol(HIP_SYMBOL(LIM), &SimParameters.prop_medium   , NMaxRegions*sizeof(HeliosphereZoneProperties_t));
        hipMemcpyToSymbol(HIP_SYMBOL(HS), &SimParameters.prop_Heliosheat, NMaxRegions*sizeof(HeliosheatProperties_t));
        // hipMemcpyToSymbol(HIP_SYMBOL(dev_Npart), &prop_launch_param.Npart, sizeof(float));
        // hipMemcpyToSymbol(HIP_SYMBOL(min_dt), &MIN_DT, sizeof(float));
        // hipMemcpyToSymbol(HIP_SYMBOL(max_dt), &MAX_DT, sizeof(float));
        // hipMemcpyToSymbol(HIP_SYMBOL(timeout), &TIMEOUT, sizeof(float));
      
        // allocate on host
        struct QuasiParticle_t host_QuasiParts = InitQuasiPart_mem(prop_launch_param.Npart, 0, VERBOSE_2);   // host initial state of propagation kernel

        // Allocate the initial variables and allocate on device
        struct QuasiParticle_t dev_QuasiParts = InitQuasiPart_mem(prop_launch_param.Npart, 1, VERBOSE_2);    // device input/output of propagation kernel

        // Period along which CR are integrated and the corresponding period indecies
        int* dev_PeriodIndexes;
        HANDLE_ERROR(hipMalloc((void**)&dev_PeriodIndexes, prop_launch_param.Npart*sizeof(int)));
        hipDeviceSynchronize();

        int* host_PeriodIndexes = (int*)malloc(prop_launch_param.Npart*sizeof(int));

        // initialize the host array
        // The particle simulated in the kernel are distributed between the initial positions using the period index
        for(int iPart=0; iPart<prop_launch_param.Npart; iPart++) {
            int PeriodIndex = floor_int(iPart*NInitPos, prop_launch_param.Npart);
            host_PeriodIndexes[iPart]    = PeriodIndex;
            host_QuasiParts.r[iPart]     = InitialPositions.r[PeriodIndex];
            host_QuasiParts.th[iPart]    = InitialPositions.th[PeriodIndex];
            host_QuasiParts.phi[iPart]   = InitialPositions.phi[PeriodIndex];
            // host_QuasiParts.R[iPart]     = InitialRigidities[floor_int(iPart*NInitRig, prop_launch_param.Npart)];
            host_QuasiParts.t_fly[iPart] = 0;
            // host_QuasiParts.alphapath[iPart] = 0;
        }

        // copy host_PeriodIndexes to dev_PeriodIndexes and free memory
        HANDLE_ERROR(hipMemcpy(dev_PeriodIndexes, host_PeriodIndexes, prop_launch_param.Npart*sizeof(int), hipMemcpyHostToDevice));

        // Recording the setting memory execution time
        if (VERBOSE){
            HANDLE_ERROR( hipEventRecord( MemorySet, 0 ) );
            HANDLE_ERROR( hipEventSynchronize( MemorySet ) );
          }      

        ////////////////////////////////////////////////////////////////
        //..... GPU perticle propagation   .............................
        ////////////////////////////////////////////////////////////////

        //  Initialization of the cycle on rigidities bins (for all the positions)
        //  Launch of the GPU propagation kernel (computing diffusion 
        //  coefficients and solving stochastic differential equations)
        //  Build the exit energy histogram


        // Cycle on rigidity bins distributing their execution between the active CPU threads
        for (int iR=gpu_id; iR<NInitRig ; iR+=NGPUs) {

            if (VERBOSE){
                HANDLE_ERROR( hipEventCreate( &Cycle_start ) );
                HANDLE_ERROR( hipEventCreate( &Cycle_step00 ) );
                HANDLE_ERROR( hipEventCreate( &Cycle_step0 ) );
                HANDLE_ERROR( hipEventCreate( &Cycle_step1 ) );
                HANDLE_ERROR( hipEventCreate( &Cycle_step2 ) );
                HANDLE_ERROR( hipEventCreate( &InitialSave ) );
                HANDLE_ERROR( hipEventCreate( &FinalSave ) );
                HANDLE_ERROR( hipEventRecord( Cycle_start, 0 ) );
              }
            
            // GPU propagation kernel execution parameters debugging
            if (VERBOSE) {
                printf("\n-- Cycle on rigidity[%d]: %.2f \n", iR , InitialRigidities[iR]);
                printf("Quasi-particles propagation kernel launched\n");
                printf("Number of quasi-particles: %d\n", prop_launch_param.Npart);
                printf("Number of blocks: %d\n", prop_launch_param.blocks);
                printf("Number of threads per block: %d\n", prop_launch_param.threads);
                printf("Number of shared memory bytes per block: %d\n", prop_launch_param.smem);
            }
            
            // Initialize the particle starting rigidities
            for(int iPart=0; iPart<prop_launch_param.Npart; iPart++) {
                host_QuasiParts.R[iPart] = InitialRigidities[iR];
            }

            // copy host initial propagation states to device quasi particle states
            HANDLE_ERROR(hipMemcpy(dev_QuasiParts.r, host_QuasiParts.r, prop_launch_param.Npart*sizeof(float), hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(dev_QuasiParts.th, host_QuasiParts.th, prop_launch_param.Npart*sizeof(float), hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(dev_QuasiParts.phi, host_QuasiParts.phi, prop_launch_param.Npart*sizeof(float), hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(dev_QuasiParts.R, host_QuasiParts.R, prop_launch_param.Npart*sizeof(float), hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(dev_QuasiParts.t_fly, host_QuasiParts.t_fly, prop_launch_param.Npart*sizeof(float), hipMemcpyHostToDevice));
            // HANDLE_ERROR(hipMemcpy(dev_QuasiParts.alphapath, host_QuasiParts.alphapath, prop_launch_param.Npart*sizeof(float), hipMemcpyHostToDevice));

            // Allocate the array for the partial rigidities maxima and final maximum
            float* dev_maxs;
            HANDLE_ERROR(hipMalloc((void **) &dev_maxs, prop_launch_param.blocks*sizeof(float)));

            #if NVIDIA_HIST
                float* dev_Rmax;
                HANDLE_ERROR(hipMalloc((void **) &dev_Rmax, 2*sizeof(float)));
                host_Rmax = (float*)malloc(2*sizeof(float));
                hipDeviceSynchronize();

            #else
                float* host_Rmax;
                host_Rmax = (float*)malloc(prop_launch_param.blocks*sizeof(float));
                hipDeviceSynchronize();
            #endif


            if (VERBOSE){
                HANDLE_ERROR( hipEventRecord( Cycle_step00, 0 ) );
                HANDLE_ERROR( hipEventSynchronize( Cycle_step00 ) );
              }        

            // Saving the initial particles parameters into a txt file for debugging
            if (INITSAVE) {
                SaveTxt_part(init_filename, prop_launch_param.Npart, host_QuasiParts, host_Rmax[0], VERBOSE_2);
            }

            if (VERBOSE){
                HANDLE_ERROR( hipEventRecord( InitialSave, 0 ) );
                HANDLE_ERROR( hipEventSynchronize( InitialSave ) );
            }


            // Heliosphere propagation kernel
            // and local max rigidity search inside the block
            HeliosphericProp<<<prop_launch_param.blocks, prop_launch_param.threads, prop_launch_param.smem>>>
            (prop_launch_param.Npart, MIN_DT, MAX_DT, TIMEOUT, dev_QuasiParts, dev_PeriodIndexes, pt, dev_RndStates, dev_maxs);

            // (taking into account the different possible block dimension template of BlockMax execution)
            /* switch (prop_launch_param.threads) {
                case 512:
                HeliosphericProp<512><<<prop_launch_param.blocks, prop_launch_param.threads, prop_launch_param.smem>>>
                (prop_launch_param.Npart, MAX_DT, TIMEOUT, dev_QuasiParts, dev_RndStates, dev_maxs);
                break;
                case 256:
                HeliosphericProp<256><<<prop_launch_param.blocks, prop_launch_param.threads, prop_launch_param.smem>>>
                (prop_launch_param.Npart, MAX_DT, TIMEOUT, dev_QuasiParts, dev_RndStates, dev_maxs);
                break;
                case 128:
                HeliosphericProp<128><<<prop_launch_param.blocks, prop_launch_param.threads, prop_launch_param.smem>>>
                (prop_launch_param.Npart, MAX_DT, TIMEOUT, dev_QuasiParts, dev_RndStates, dev_maxs);
                break;
                case 64:
                HeliosphericProp< 64><<<prop_launch_param.blocks, prop_launch_param.threads, prop_launch_param.smem>>>
                (prop_launch_param.Npart, MAX_DT, TIMEOUT, dev_QuasiParts, dev_RndStates, dev_maxs);
                break;
                case 32:
                HeliosphericProp< 32><<<prop_launch_param.blocks, prop_launch_param.threads, prop_launch_param.smem>>>
                (prop_launch_param.Npart, MAX_DT, TIMEOUT, dev_QuasiParts, dev_RndStates, dev_maxs);
                break;
            } */
            
            hipDeviceSynchronize();

            if (VERBOSE) {
                HANDLE_ERROR( hipEventRecord( Cycle_step0, 0 ) );
                HANDLE_ERROR( hipEventSynchronize( Cycle_step0 ) );
            }        
        
            #if NVIDIA_HIST
                // Global max rigidity search from partial maxima
                unsigned int GridMax_threads = ceil_int(prop_launch_param.blocks/2, (device_prop.warpSize))*(device_prop.warpSize);
                GridMax<<<2, 2*GridMax_threads, GridMax_threads*sizeof(float)>>>(prop_launch_param.blocks, dev_maxs, dev_Rmax);
                
                // (taking into account the different possible block dimension template of GridMax execution)
                /* switch (prop_launch_param.blocks) {
                    case 512:
                        GridMax<512><<<1, prop_launch_param.blocks, prop_launch_param.blocks*sizeof(float)>>>(dev_maxs, dev_Rmax);
                        break;
                    case 256:
                        GridMax<256><<<1, prop_launch_param.blocks, prop_launch_param.blocks*sizeof(float)>>>(dev_maxs, dev_Rmax);
                        break;
                    case 128:
                        GridMax<128><<<1, prop_launch_param.blocks, prop_launch_param.blocks*sizeof(float)>>>(dev_maxs, dev_Rmax);
                        break;
                    case 64:
                        GridMax< 64><<<1, prop_launch_param.blocks, prop_launch_param.blocks*sizeof(float)>>>(dev_maxs, dev_Rmax);
                        break;
                    case 32:
                        GridMax< 32><<<1, prop_launch_param.blocks, prop_launch_param.blocks*sizeof(float)>>>(dev_maxs, dev_Rmax);
                        break;
                } */

                // Copy the final maximum rigidity to host and free partial maxima array memory
                HANDLE_ERROR(hipMemcpy(host_Rmax, dev_Rmax, 2*sizeof(float), hipMemcpyDeviceToHost));
                hipFree(dev_Rmax);

                // Finalization of the maximum rigidity search on CPU
                if (host_Rmax[0]<host_Rmax[1]) host_Rmax[0] = host_Rmax[1];

            #else

                hipMemcpy(host_Rmax, dev_maxs, prop_launch_param.blocks*sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();    

                if (VERBOSE_2){
                    fprintf(stdout,"--- Max values: ");
                    for (int itemp=0; itemp<prop_launch_param.blocks; itemp++) {
                        fprintf(stdout,"%.2f ", host_Rmax[itemp]);
                    }
                    fprintf(stdout,"\n");
                    fprintf(stdout,"--- EMin = %.3f Emax = %.3f \n",InitialRigidities[iR], host_Rmax[0]);
                }

                // ->then finalize on CPU
                for (int itemp=1; itemp<prop_launch_param.blocks; itemp++) {
                    if (host_Rmax[0] < host_Rmax[itemp]) {
                        host_Rmax[0] = host_Rmax[itemp];
                    }
                }

                if (host_Rmax[0]<SimParameters.Tcentr[iR]){
                    printf("PROBLEMA: the max exiting energy is bigger than initial one\n");
                    continue;
                }
            #endif

            hipFree(dev_maxs);

            if (VERBOSE){
                HANDLE_ERROR( hipEventRecord( Cycle_step1, 0 ) );
                HANDLE_ERROR( hipEventSynchronize( Cycle_step1 ) );
            }

            if (FINALSAVE) {    
                // host final states for specific energy
                struct QuasiParticle_t host_final_QuasiParts = InitQuasiPart_mem(prop_launch_param.Npart, 0, VERBOSE_2);

                // copy device final propagation states to host quasi particle states
                HANDLE_ERROR(hipMemcpy(host_final_QuasiParts.r, dev_QuasiParts.r, prop_launch_param.Npart*sizeof(float), hipMemcpyDeviceToHost));
                HANDLE_ERROR(hipMemcpy(host_final_QuasiParts.th, dev_QuasiParts.th, prop_launch_param.Npart*sizeof(float), hipMemcpyDeviceToHost));
                HANDLE_ERROR(hipMemcpy(host_final_QuasiParts.phi, dev_QuasiParts.phi, prop_launch_param.Npart*sizeof(float), hipMemcpyDeviceToHost));
                HANDLE_ERROR(hipMemcpy(host_final_QuasiParts.R, dev_QuasiParts.R, prop_launch_param.Npart*sizeof(float), hipMemcpyDeviceToHost));
                HANDLE_ERROR(hipMemcpy(host_final_QuasiParts.t_fly, dev_QuasiParts.t_fly, prop_launch_param.Npart*sizeof(float), hipMemcpyDeviceToHost));
                // HANDLE_ERROR(hipMemcpy(host_final_QuasiParts.alphapath, dev_QuasiParts.alphapath, prop_launch_param.Npart*sizeof(float), hipMemcpyDeviceToHost));

                // Saving the propagation output into a txt file
                SaveTxt_part(final_filename, prop_launch_param.Npart, host_final_QuasiParts, host_Rmax[0], VERBOSE_2);

                // Free the host particle variable for the energy on which the cycle is running
                free(host_final_QuasiParts.r);
                free(host_final_QuasiParts.th);
                free(host_final_QuasiParts.phi);
                free(host_final_QuasiParts.R);
                free(host_final_QuasiParts.t_fly);
                // free(host_final_QuasiParts.alphapath);
            } 
            
            if (VERBOSE){
                HANDLE_ERROR( hipEventRecord( FinalSave, 0 ) );
                HANDLE_ERROR( hipEventSynchronize( FinalSave ) );
            }

            // Definition of histogram binning as a fraction of the bin border (DeltaT=T*RelativeBinAmplitude)
            float DeltaLogR= log10(1.+Rigidity(RelativeBinAmplitude, pt));
            float LogBin0_lowEdge = log10(InitialRigidities[iR])-(DeltaLogR/2.);
            float Bin0_lowEdge = pow(10, LogBin0_lowEdge );                     // first LowEdge Bin

            Results[iR].Nbins           = ceilf(log10(host_Rmax[0]/Bin0_lowEdge)/DeltaLogR);
            Results[iR].LogBin0_lowEdge = LogBin0_lowEdge;
            Results[iR].DeltaLogR       = DeltaLogR;

            free(host_Rmax);

            // .. save to histogram ..........................................
            // Partial block histogram allocation
            float* dev_partialHistos;
            HANDLE_ERROR(hipMalloc((void**) &dev_partialHistos, Results[iR].Nbins*prop_launch_param.blocks*sizeof(float)));

            // Final merged histogram allocation
            Results[iR].BoundaryDistribution = (float*)malloc(Results[iR].Nbins*sizeof(float));
            float* dev_Histo;
            HANDLE_ERROR(hipMalloc((void **)&dev_Histo, Results[iR].Nbins*sizeof(float)));

            #if NVIDIA_HIST
                
                // Partial histogram atomoic sum (exploiting the shared memory)
                Rhistogram_atomic<<<prop_launch_param.blocks, prop_launch_param.threads, Results[iR].Nbins*sizeof(int)>>>(dev_QuasiParts.R, LogBin0_lowEdge, DeltaLogR , Results[iR].Nbins, prop_launch_param.Npart,  dev_partialHistos);
                
                /* int* host_partialHistos = (int*)malloc(Results[iR].Nbins*prop_launch_param.blocks*sizeof(int));
                hipMemcpy(host_partialHistos, dev_partialHistos, Results[iR].Nbins*prop_launch_param.blocks*sizeof(int), hipMemcpyDeviceToHost);
                printf("dev_partialHistos: \n");
                for (int i=0; i<Results[iR].Nbins*prop_launch_param.blocks; i++) {
                    printf("%d ", host_partialHistos[i]);
                } */

                hipDeviceSynchronize();
                
                // Merge of the partial histograms and copy to the host
                TotalHisto<<<Results[iR].Nbins, prop_launch_param.blocks/2, (prop_launch_param.blocks/2)*sizeof(int)>>>(dev_partialHistos, Results[iR].Nbins, prop_launch_param.blocks, dev_Histo);
                hipMemcpy(Results[iR].BoundaryDistribution, dev_Histo, Results[iR].Nbins*sizeof(float),hipMemcpyDeviceToHost);

            #else

                int *dev_Nfailed;
                HANDLE_ERROR(hipMalloc((void **) &dev_Nfailed, sizeof(int))) ;
                hipMemset(dev_Nfailed, 0, sizeof(int));
                
                // Partial histogram atomoic sum on GPU
                histogram_atomic<<<prop_launch_param.blocks, prop_launch_param.threads>>>(dev_QuasiParts.R,  LogBin0_lowEdge, DeltaLogR, Results[iR].Nbins, prop_launch_param.Npart,  dev_partialHistos, dev_Nfailed);
                
                // Failed quasi-particle propagation count
                int Nfailed=0;
                hipMemcpy(&Nfailed, dev_Nfailed, sizeof(int),hipMemcpyDeviceToHost);
                Results[iR].Nregistered = prop_launch_param.Npart-Nfailed;
                
                if (VERBOSE){
                    fprintf(stdout,"-- Eventi computati : %lu \n", prop_launch_param.Npart);
                    fprintf(stdout,"-- Eventi falliti   : %d \n", Nfailed);
                    fprintf(stdout,"-- Eventi registrati: %lu \n", Results[iR].Nregistered);
                }
                hipDeviceSynchronize();

                int histo_Nblocchi = ceil_int(Results[iR].Nbins, prop_launch_param.threads);
                
                // Merge of the partial histograms and copy to the host
                histogram_accum<<<histo_Nblocchi, prop_launch_param.threads>>>(dev_partialHistos, Results[iR].Nbins, prop_launch_param.blocks, dev_Histo);
                hipMemcpy(Results[iR].BoundaryDistribution, dev_Histo, Results[iR].Nbins*sizeof(float),hipMemcpyDeviceToHost);

                hipFree(dev_Nfailed);
            #endif

            hipFree(dev_partialHistos);
            hipFree(dev_Histo);


            // ANNOTATION THE ONLY MEMCOPY NEEDED FROM DEVICE TO HOST ARE THE FINAL RESULTS (ALIAS THE ENERGY FINAL HISTOGRAM AND PARTICLE EXIT RESULTS)

            // .. ............................................................
            if (VERBOSE){
                HANDLE_ERROR( hipEventRecord( Cycle_step2, 0 ) );
                HANDLE_ERROR( hipEventSynchronize( Cycle_step2 ) );
                float   Enl00,Enl0,Enl1,Enl2, EnlIn, EnlFin;
                HANDLE_ERROR( hipEventElapsedTime( &Enl00,
                                                    Cycle_start, Cycle_step00 ) );
                HANDLE_ERROR( hipEventElapsedTime( &EnlIn,
                                                    Cycle_step00, InitialSave ) );
                HANDLE_ERROR( hipEventElapsedTime( &Enl0,
                                                    InitialSave, Cycle_step0 ) );
                HANDLE_ERROR( hipEventElapsedTime( &Enl1,
                                                    Cycle_step0, Cycle_step1 ) );
                HANDLE_ERROR( hipEventElapsedTime( &EnlFin,
                                                    Cycle_step1, FinalSave ) );
                HANDLE_ERROR( hipEventElapsedTime( &Enl2,
                                                    FinalSave, Cycle_step2 ) );
                printf( "-- Init              :  %3.2f ms \n", Enl00 );                                         
                printf( "-- Save initial state:  %3.2f ms \n", EnlIn );                                         
                printf( "-- Propagation phase :  %3.2f ms \n", Enl0 );
                printf( "-- Find Max          :  %3.2f ms \n", Enl1 );
                printf( "-- Save final state  :  %3.2f ms \n", EnlFin );
                printf( "-- Binning           :  %3.2f ms \n", Enl2 );    
                HANDLE_ERROR( hipEventDestroy( Cycle_start ) ); 
                HANDLE_ERROR( hipEventDestroy( Cycle_step00 ) );
                HANDLE_ERROR( hipEventDestroy( InitialSave ) );
                HANDLE_ERROR( hipEventDestroy( Cycle_step0 ) );
                HANDLE_ERROR( hipEventDestroy( Cycle_step1 ) );
                HANDLE_ERROR( hipEventDestroy( FinalSave ) );
                HANDLE_ERROR( hipEventDestroy( Cycle_step2 ) );
            }
        }
        // end of the cycle on the rigidities

        if (VERBOSE){
            HANDLE_ERROR( hipEventRecord( stop, 0 ) );
            HANDLE_ERROR( hipEventSynchronize( stop ) );
        }
        // Execution Time
        if (VERBOSE){
            float   elapsedTime,firstStep,memset;
            HANDLE_ERROR( hipEventElapsedTime( &memset,
                                                    start, MemorySet ) );
            HANDLE_ERROR( hipEventElapsedTime( &firstStep,
                                                    start, Randomstep ) );
            HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                                    start, stop ) );  
            printf( "Time to Set Memory:  %3.1f ms \n", memset );
            printf( "Time to create Rnd:  %3.1f ms (delta = %3.1f)\n", firstStep, firstStep-memset );                                     
            printf( "Time to execute   :  %3.1f ms (delta = %3.1f)\n", elapsedTime, elapsedTime-firstStep);
    
        }
      

        ////////////////////////////////////////////////////////////////
        //..... Exit results saving   ..................................
        ////////////////////////////////////////////////////////////////

        //  Save the summary histogram
        //  Free the dynamic memory

        // Save the rigidity histograms to txt file
        for (int iR=0; iR<NInitRig; iR++) {
            SaveTxt_histo(histo_filename, Results[iR].Nbins, Results[iR], VERBOSE_2);
        }

        /* save results to file .dat */
        #if HELMOD_LOAD
            FILE * pFile_Matrix=NULL;
            char RAWMatrix_name[MaxCharinFileName];
            sprintf(RAWMatrix_name,"%s_matrix_%lu.dat", SimParameters.output_file_name, (unsigned long int)getpid());

            if (VERBOSE) fprintf(stdout,"Writing Output File: %s \n", RAWMatrix_name);
            pFile_Matrix = fopen (RAWMatrix_name, "w");
            
            if (pFile_Matrix==NULL) {
                fprintf(stderr, ERR_NoOutputFile);
                fprintf(stderr, "Writing to StandardOutput instead\n");
                pFile_Matrix = stdout;
            }

            fprintf(pFile_Matrix, "# COSMICA \n");
            if (VERBOSE) fprintf(pFile_Matrix, "# Number of Input energies;\n");
            fprintf(pFile_Matrix, "%d \n", SimParameters.NT);

            for (int itemp=0; itemp<SimParameters.NT; itemp++) {
                if (VERBOSE) {
                    fprintf(pFile_Matrix,"######  Bin %d \n", itemp);
                    fprintf(pFile_Matrix,"# Rgen, Npart Gen., Npart Registered, Nbin output, log10(lower edge bin 0), Bin amplitude (in log scale)\n");
                }
                
                fprintf(pFile_Matrix,"%f %lu %lu %d %f %f \n",SimParameters.Tcentr[itemp],
                                                            SimParameters.Npart,
                                                                          Results[itemp].Nregistered,
                                                                          Results[itemp].Nbins,
                                                                          Results[itemp].LogBin0_lowEdge,
                                                                          Results[itemp].DeltaLogR);                   
                if (VERBOSE) fprintf(pFile_Matrix, "# output distribution \n");
        
                for (int itNB=0; itNB<Results[itemp].Nbins; itNB++) {
                    fprintf(pFile_Matrix, "%e ", Results[itemp].BoundaryDistribution[itNB]);
                }

                fprintf(pFile_Matrix,"\n");
                fprintf(pFile_Matrix,"#\n"); // <--- dummy line to separate results
            }

            fflush(pFile_Matrix);
            fclose(pFile_Matrix);
        #endif

        // Free the host and device memory
        hipFree(dev_PeriodIndexes);

        free(host_QuasiParts.r);
        free(host_QuasiParts.th);
        free(host_QuasiParts.phi);
        free(host_QuasiParts.R);
        free(host_QuasiParts.t_fly);
        // free(host_QuasiParts.alphapath);

        hipFree(dev_RndStates);
        hipFree(dev_QuasiParts.r);
        hipFree(dev_QuasiParts.th);
        hipFree(dev_QuasiParts.phi);
        hipFree(dev_QuasiParts.R);
        hipFree(dev_QuasiParts.t_fly);
        // hipFree(dev_QuasiParts.alphapath);

        free(host_PeriodIndexes);

        if (VERBOSE) {
            HANDLE_ERROR( hipEventDestroy( start ) );
            HANDLE_ERROR( hipEventDestroy( Randomstep ) );
            HANDLE_ERROR( hipEventDestroy( stop ) );
        }      
    }
    // end of the multiple CPU thread pragma

    // Free of the initial simulation variables
    free(InitialPositions.r);
    free(InitialPositions.th);
    free(InitialPositions.phi);
    free(InitialRigidities);


    free(GPUs_profile);

    if (VERBOSE) {
        // -- Save end time of simulation into log file
        time_t tim =time(NULL);
        struct tm *local = localtime(&tim);
        printf("\nSimulation end at: %s  \n",asctime(local));
    }


    return EXIT_SUCCESS;

}